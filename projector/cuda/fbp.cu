#include "hip/hip_runtime.h"
#include "fbp.h"
#include "cudaMath.h"
#include "siddon.h"

#include <math.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <sstream>
#include <iostream>

using namespace std;

__global__ void ComplexMultiply2D(hipfftComplex* res, const hipfftComplex* op1, const hipfftComplex* op2, size_t nx, size_t ny)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= nx || iy >= ny)
    {
        return;
    }

    size_t ind = iy * nx + ix;
    hipfftComplex val1 = op1[ind];
    hipfftComplex val2 = op2[ind];
    hipfftComplex val;

    val.x = val1.x * val2.x - val1.y * val2.y;
    val.y = val1.x * val2.y + val1.y * val2.x;

    res[ind] = val;
}

// The pcuPrjPad is of size [nview, nuPad]
__global__ void CopyPrjToPad(float* pcuPrjPad, const float* pcuPrj, int iv, 
    size_t nu, size_t nuPad, size_t nv, size_t nview)
{
    int iu = blockIdx.x * blockDim.x + threadIdx.x;
    int iview = blockIdx.y * blockDim.y + threadIdx.y;

    if (iu >= nu || iview >= nview)
    {
        return;
    }

    pcuPrjPad[iview * nuPad + iu] = pcuPrj[iview * nu * nv + iv * nu + iu];
}

__global__ void CopyPadToPrj(const float* pcuPrjPad, float* pcuPrj, int iv,
    size_t nu, size_t nuPad, size_t nv, size_t nview)
{
    int iu = blockIdx.x * blockDim.x + threadIdx.x;
    int iview = blockIdx.y * blockDim.y + threadIdx.y;

    if (iu >= nu || iview >= nview)
    {
        return;
    }

    pcuPrj[iview * nu * nv + iv * nu + iu] = pcuPrjPad[iview * nuPad + iu];
}

void GetRamp(hipfftComplex* pcuFreqKernel, size_t nu, size_t nview, float da, int filterType, hipStream_t& stream,
    bool isEqualSpace = false)
{
    int filterLen = 2 * nu - 1;

    // fft plan
    hipfftHandle plan = 0;
    hipfftHandle planR2C = 0;
    hipfftComplex* pcuRamp = NULL;
    hipfftReal *pcuRealKernel = NULL;
    hipfftComplex* pRamp = NULL;
    hipfftComplex* pWindow = NULL;
    hipfftReal* pRealKernel = NULL;

    try
    {
        if (HIPFFT_SUCCESS != hipfftPlan1d(&plan, filterLen, HIPFFT_C2C, 1))
        {
            throw std::runtime_error("hipfftPlan1d failure in GetRamp()");
        }
    
        if (HIPFFT_SUCCESS != hipfftSetStream(plan, stream))
        {
            throw std::runtime_error("cudaSetStream failure in GetRamp()");
        }
    
        // RL kernel
        if (hipSuccess != hipMalloc(&pcuRamp, sizeof(hipfftComplex) * filterLen))
        {
            throw std::runtime_error("pcuRamp allocation error in GetRamp()");
        }
        pRamp = new hipfftComplex [filterLen];
        if (isEqualSpace)
        {
            // equispace
            for (int i = 0; i < filterLen; i++)
            {
                int k = i - (nu - 1);
                if (k == 0)
                {
                    pRamp[i].x = 1 / (4 * da * da);
                }
                else if (k % 2 != 0)
                {
                    pRamp[i].x = -1 / (PI * PI * k * k * da * da);
                }
                else
                {
                    pRamp[i].x = 0;
                }
                pRamp[i].y = 0;
            }
        }
        else
        {
            // equiangular
            for (int i = 0; i < filterLen; i++)
            {
                int k = i - (nu - 1);
                if (k == 0)
                {
                    pRamp[i].x = 1 / (4 * da * da);
                }
                else if (k % 2 != 0)
                {
                    pRamp[i].x = -1 / (PI * PI * sinf(k*da) * sinf(k*da));
                }
                else
                {
                    pRamp[i].x = 0;
                }
                pRamp[i].y = 0;
            }
        }
    
        hipMemcpyAsync(pcuRamp, pRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyHostToDevice, stream);
        hipfftExecC2C(plan, pcuRamp, pcuRamp, HIPFFT_FORWARD);
        hipMemcpyAsync(pRamp, pcuRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyDeviceToHost, stream);
    
        // weighting window in frequency domain
        pWindow = new hipfftComplex [filterLen];
        switch(filterType)
        {
        case FILTER_HAMMING:
            // Hamming
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = 0.54f + 0.46f * cosf(2 * PI * i / (float)(filterLen-1));
                pWindow[i].y = 0;
            }
            break;
        case FILTER_HANN:
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = 0.5f + 0.5f * cosf(2 * PI * i / (float)(filterLen-1));
                pWindow[i].y = 0;
            }
            break;
        case FILTER_COSINE:
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = abs(cosf(PI * i / (float)(filterLen-1)));
                pWindow[i].y = 0;
            }
            break;
        default:
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = 1;
                pWindow[i].y = 0;
            }
        }
    
        // Apply window on the filter
        for (int i = 0; i < filterLen; i++)
        {
            float real = pRamp[i].x * pWindow[i].x - pRamp[i].y * pWindow[i].y;
            float imag = pRamp[i].x * pWindow[i].y + pRamp[i].y * pWindow[i].x;
            pRamp[i].x = real;
            pRamp[i].y = imag;
        }
    
        hipMemcpyAsync(pcuRamp, pRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyHostToDevice, stream);
        hipfftExecC2C(plan, pcuRamp, pcuRamp, HIPFFT_BACKWARD);
        hipMemcpyAsync(pRamp, pcuRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyDeviceToHost, stream);

        if (hipSuccess != hipMalloc(&pcuRealKernel, sizeof(hipfftReal) * filterLen))
        {
            throw std::runtime_error("pRealKernel allocation error in getRampEA");
        }
        hipfftReal* pRealKernel = new hipfftReal [filterLen];
        for (int i = 0; i < filterLen; i++)
        {
            pRealKernel[i] = pRamp[i].x / filterLen;
        }
        hipMemcpyAsync(pcuRealKernel, pRealKernel, sizeof(hipfftReal) * filterLen, hipMemcpyHostToDevice, stream);
    
        hipfftHandle planR2C;
        hipfftPlan1d(&planR2C, filterLen, HIPFFT_R2C, 1);
        hipfftSetStream(planR2C, stream);
        hipfftExecR2C(planR2C, pcuRealKernel, pcuFreqKernel);
        for (int i = 1; i < nview; i++)
        {
            hipMemcpyAsync(pcuFreqKernel + i * nu, pcuFreqKernel, sizeof(hipfftComplex) * nu, hipMemcpyDeviceToDevice, stream);
        }

    }
    catch (exception &e)
    {
        if (pRamp != NULL) delete [] pRamp;
        if (pWindow != NULL) delete [] pWindow;
        if (pRealKernel != NULL) delete [] pRealKernel;
        if (pcuRamp != NULL) hipFree(pcuRamp);
        if (pcuRealKernel != NULL) hipFree(pcuRealKernel);
        if (plan != 0) hipfftDestroy(plan);
        if (planR2C != 0) hipfftDestroy(planR2C);

        ostringstream oss;
        oss << "GetRamp() failed: " << e.what()
                << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
        throw runtime_error(oss.str().c_str());
    }

    if (pRamp != NULL) delete [] pRamp;
    if (pWindow != NULL) delete [] pWindow;
    if (pRealKernel != NULL) delete [] pRealKernel;
    if (pcuRamp != NULL) hipFree(pcuRamp);
    if (pcuRealKernel != NULL) hipFree(pcuRealKernel);
    if (plan != 0) hipfftDestroy(plan);
    if (planR2C != 0) hipfftDestroy(planR2C);

}

void fbpFan::Filter(float* pcuFPrj, const float* pcuPrj)
{
    bool isEqualSpace = false;

    // the filter is carried out for each different v
    int filterLen = nu * 2 - 1;
    float* pcuPrjPad = NULL;
    hipfftComplex* pcuFreqPrj = NULL;
    hipfftComplex* pcuFilter = NULL;
    float* pw = NULL;
    float* pcuw = NULL;
    hipfftHandle plan = 0;
    hipfftHandle planInverse = 0;

    try
    {
        // projection
        if (hipSuccess != hipMalloc(&pcuPrjPad, sizeof(float) * filterLen * nview))
        {
            throw std::runtime_error("pcuPrjPad allocation failure in fan3D::Filter");
        }

        // freq projection
        if (hipSuccess != hipMalloc(&pcuFreqPrj, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFreqPrj allocation failure in fan3D::Filter");
        }

        // filter
        if (hipSuccess != hipMalloc(&pcuFilter, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFilter allocation failure in fan3D::Filter");
        }
        GetRamp(pcuFilter, nu, nview, du, typeProjector, m_stream, isEqualSpace);

        // Get projection weighting
        pw = new float [nu];
        if (isEqualSpace)
        {
            for (int i = 0; i < nu; i++)
            {
                float u = ((i - (nu - 1) / 2.f) - off_u) * du;
                pw[i] = dsd / sqrtf(dsd * dsd + u * u);
            }
        }
        else
        {
            for (int i = 0; i < nu; i++)
            {
                float angle = ((i - (nu - 1) / 2.f) - off_u) * du;
                pw[i] = cosf(angle);
            }
        }

        
        if (hipSuccess != hipMalloc(&pcuw, sizeof(float) * nu * nview))
        {
            throw std::runtime_error("pcuw allocation failure in fan3D::Filter");
        }
        hipMemcpyAsync(pcuw, pw, sizeof(float) * nu, hipMemcpyHostToDevice, m_stream);
        for (int i = 1; i < nview; i++)
        {
            hipMemcpyAsync(pcuw + i * nu, pcuw, sizeof(float) * nu, hipMemcpyDeviceToDevice, m_stream);
        }

        // fft plan
        if (HIPFFT_SUCCESS != hipfftPlanMany(&plan, 1, &filterLen, NULL, 1, filterLen, NULL, 1, nu, HIPFFT_R2C, nview))
        {
            throw std::runtime_error("fft plan error in fan3D::Filter");
        }
        hipfftSetStream(plan, m_stream);
        
        if (HIPFFT_SUCCESS != hipfftPlanMany(&planInverse, 1, &filterLen, NULL, 1, nu, NULL, 1, filterLen, HIPFFT_C2R, nview))
        {
            throw std::runtime_error("ifft plan error in fan3D::Filter");
        }
        hipfftSetStream(plan, m_stream);

        // kernel threads and blocks
        dim3 threads(32, 16, 1);
        dim3 blocks(ceilf(nu / (float)threads.x), ceilf(nview / (float)threads.y), 1);
        float scale;
        if (isEqualSpace)
        {
            scale = PI / nview * du * dsd / dso / filterLen;
        }
        else
        {
            scale = PI / nview * du / dso / filterLen;
        }

        for (int ib = 0; ib < nBatches; ib++)
        {
            for (int iv = 0; iv < nv; iv++)
            {
                hipMemsetAsync(pcuPrjPad, 0, sizeof(float) * filterLen * nview, m_stream);
                CopyPrjToPad<<<blocks, threads, 0, m_stream>>>(pcuPrjPad,
                        pcuPrj + ib * nu * nv * nview,
                        iv, nu, filterLen, nv, nview);

                // pre weighting
                Multiply2D<<<blocks, threads, 0, m_stream>>>(pcuPrjPad, pcuPrjPad, pcuw,
                        nu, nview, filterLen, filterLen, nu);
                hipDeviceSynchronize();

                hipfftExecR2C(plan, pcuPrjPad, pcuFreqPrj);
                ComplexMultiply2D<<<blocks, threads, 0, m_stream>>>(pcuFreqPrj, pcuFreqPrj, pcuFilter, nu, nview);
                hipDeviceSynchronize();
                hipfftExecC2R(planInverse, pcuFreqPrj, pcuPrjPad);

                // post scaling
                Scale2D<<<blocks, threads, 0, m_stream>>>(pcuPrjPad + nu - 1, pcuPrjPad + nu - 1,
                        scale, nu, nview, filterLen, filterLen);

                CopyPadToPrj<<<blocks, threads, 0, m_stream>>>(pcuPrjPad + nu - 1,
                        pcuFPrj + ib * nu * nv * nview,
                        iv, nu, filterLen, nv, nview);
            }
        }
    }
    catch (exception &e)
    {
        if (plan != 0) hipfftDestroy(plan);
        if (planInverse != 0) hipfftDestroy(planInverse);
        if (pcuPrjPad != NULL) hipFree(pcuPrjPad);
        if (pcuFreqPrj != NULL) hipFree(pcuFreqPrj);
        if (pcuFilter != NULL) hipFree(pcuFilter);
        if (pcuw != NULL) hipFree(pcuw);
        if (pw != NULL) delete [] pw;

        ostringstream oss;
        oss << "fbpFan::Filter() failed: " << e.what()
                << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
        throw runtime_error(oss.str().c_str());
    }

    if (plan != 0) hipfftDestroy(plan);
    if (planInverse != 0) hipfftDestroy(planInverse);
    if (pcuPrjPad != NULL) hipFree(pcuPrjPad);
    if (pcuFreqPrj != NULL) hipFree(pcuFreqPrj);
    if (pcuFilter != NULL) hipFree(pcuFilter);
    if (pcuw != NULL) hipFree(pcuw);
    if (pw != NULL) delete [] pw;

}

extern "C" int cfbpFanFilter(float* pFPrj, const float* pPrj,
    int nBatches, 
    size_t nu, size_t nv, size_t nview, float da, float dv, float off_a, float off_v,
    float dsd, float dso, int typeFilter = 0)
{
    fbpFan filter;
    filter.Setup(nBatches, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        nu, nv, nview, da, dv, off_a, off_v, dsd, dso, typeFilter);
    float* pcuFPrj = NULL;
    float* pcuPrj = NULL;

    try
    {
        if (hipSuccess != hipMalloc(&pcuFPrj, sizeof(float) * nBatches * nu * nv * nview))
        {
            throw runtime_error("pcuFPrj allocation failed");
        }
        if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nBatches * nu * nv * nview))
        {
            throw runtime_error("pcuPrj allocation failed");
        }
        hipMemset(pcuFPrj, 0, sizeof(float) * nBatches * nu * nv * nview);
        hipMemcpy(pcuPrj, pPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyHostToDevice);

        filter.Filter(pcuFPrj, pcuPrj);

        hipMemcpy(pFPrj, pcuFPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyDeviceToHost);
    }
    catch (exception &e)
    {
        ostringstream oss;
        oss << "cFilterFanFilter() failed: " << e.what()
                << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
    }

    if (pcuFPrj != NULL) hipFree(pcuFPrj);
    if (pcuPrj != NULL) hipFree(pcuPrj);

    return hipGetLastError();

}

/**************************
Backprojection
**************************/

const static int nzBatch = 5;
__global__ void bpFanKernel3D(float* pImg, const float* prj, const float* pDeg,
    size_t nview, const Grid grid, const Detector det, float dsd, float dso, bool isFBP)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int izBatch = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= grid.nx || iy >= grid.ny || izBatch * nzBatch >= grid.nz)
	{
		return;
	}

    // the image coordinates here has the lower left corner of the first pixel defined as (0,0,0)
    // because (ix,iy,iz) are based on the centers of the pixels, so an offset of 0.5 should be added. 
    register float3 pt = ImgToPhysics(make_float3(ix + 0.5f, iy + 0.5f, izBatch * nzBatch + 0.5f), grid);

	register float val[nzBatch] = {0};
	register float cosDeg, sinDeg, rx, ry, pu, pv, a, dist;
	for (int iview = 0; iview < nview; iview++)
	{
		cosDeg = __cosf(pDeg[iview]);
		sinDeg = __sinf(pDeg[iview]);
		rx =  pt.x*cosDeg + pt.y*sinDeg;
		ry = -pt.x*sinDeg + pt.y*cosDeg;
		a = atanf(rx/(ry+dso));
		if (isFBP)
		{
			dist = dso*dso / (rx*rx + (dso+ry)*(dso+ry));
		}
		else
		{
			float sin_a = fabs(__sinf(a));
			if (sin_a > 1e-6f)
			{
				dist = fminf(grid.dy / __cosf(a), grid.dx / sin_a);
			}
			else
			{
				dist = grid.dy / __cosf(a);
			}
		}

		pu = -(a/det.du + det.off_u) + (det.nu - 1.0f) / 2.0f;

#pragma unroll
		for (int iz = 0; iz < nzBatch; iz++)
		{
			pv = (pt.z + iz * grid.dz) / det.dv + det.off_v + (det.nv - 1.0f) / 2.f;

            // val[iz] = dist;
            val[iz] += InterpolateXY(prj, pu, pv, iview, det.nu, det.nv, nview, true) * dist;
		}

	}
#pragma unroll
	for (int iz = 0; iz < nzBatch; iz++)
	{
		if (iz + izBatch * nzBatch < grid.nz)
		{
			pImg[(iz + izBatch * nzBatch) * grid.nx * grid.ny + iy * grid.nx + ix] += val[iz];
		}
	}

}

void fbpFan::Backprojection(float* pcuImg, const float* pcuPrj, const float* pcuDeg)
{
    dim3 threads(32, 16, 1);
	dim3 blocks(ceilf(nx / (float)threads.x), ceilf(ny / (float)threads.y), ceilf(nz / (float)nzBatch));

	for (int ib = 0; ib < nBatches; ib++)
	{
        bpFanKernel3D<<<blocks, threads, 0, m_stream>>>(
            pcuImg + ib * nx * ny * nz, pcuPrj + ib * nu * nv * nview, pcuDeg, 
            nview,
            MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz),
            MakeDetector(nu, nv, du, dv, off_u, off_v),
            dsd, dso, true);
        hipDeviceSynchronize();
	}
}

extern "C" int cfbpFanBackprojection(float* pImg, const float* pPrj, const float* pDeg,
    size_t nBatches, 
    size_t nx, size_t ny, size_t nz, float dx, float dy, float dz, float cx, float cy, float cz, 
    size_t nu, size_t nv, size_t nview, float da, float dv, float off_a, float off_v,
    float dsd, float dso)
{
    fbpFan projector;
    projector.Setup(nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
            nu, nv, nview, da, dv, off_a, off_v, dsd, dso);

    float* pcuImg = NULL;
    float* pcuPrj = NULL;
    float* pcuDeg = NULL;
    try
    {
        if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nBatches * nx * ny * nz))
        {
            throw runtime_error("pcuImg allocation failed");
        }

        if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nBatches * nu * nview * nv))
        {
            throw runtime_error("pcuPrj allocation failed");
        }

        if (hipSuccess != hipMalloc(&pcuDeg, sizeof(float) * nview))
        {
            throw runtime_error("pcuDeg allocation failed");
        }

        hipMemcpy(pcuPrj, pPrj, sizeof(float) * nBatches * nu * nview * nv, hipMemcpyHostToDevice);
        hipMemcpy(pcuDeg, pDeg, sizeof(float) * nview, hipMemcpyHostToDevice);
        hipMemset(pcuImg, 0, sizeof(float) * nBatches * nx * ny * nz);

        projector.Backprojection(pcuImg, pcuPrj, pcuDeg);
        hipMemcpy(pImg, pcuImg, sizeof(float) * nBatches * nx * ny * nz, hipMemcpyDeviceToHost);

    }
    catch (exception &e)
    {
        ostringstream oss;
        oss << "cfbpFanBackprojection() failed: " << e.what()
                << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
    }

    if (pcuImg != NULL) hipFree(pcuImg);
    if (pcuPrj != NULL) hipFree(pcuPrj);
    if (pcuDeg != NULL) hipFree(pcuDeg);

    return hipGetLastError();
}