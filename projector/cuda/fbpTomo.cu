#include "hip/hip_runtime.h"
#include "fbp.h"
#include "cudaMath.h"
#include "siddon.h"
#include "fbpMath.h"

#include <math.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <sstream>
#include <iostream>

using namespace std;

void fbpTomo::Setup(
    int nBatches, 
    size_t nx,
    size_t ny,
    size_t nz,
    float dx,
    float dy,
    float dz,
    float cx,
    float cy,
    float cz,
    size_t nu,
    size_t nv,
    size_t nview,
    float du,
    float dv,
    float off_u,
    float off_v,
    float dsd,
    float dso,
    int typeProjector,
    float cutoffX,
    float cutoffZ
)
{
    Projector::Setup(
        nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
        nu, nv, nview, du, dv, off_u, off_v, dsd, dso, typeProjector
    );

    this->cutoffX = cutoffX;
    this->cutoffZ = cutoffZ;
}

float GetFrequency(int i, int filterLen, float du, float dx, float angleWeight, float cutoff)
{
    float fu, wx;
    if (i > (filterLen + 1) / 2)
    {
        fu = (i - filterLen) / float(filterLen) / du;
    }
    else
    {
        fu = i / float(filterLen) / du;
    }
    wx = fu * angleWeight * dx * 2;
    if (wx > cutoff)
    {
        wx = cutoff;
    }
    
    return wx;
}

// always equispace
void GetRampTomo(
    hipfftComplex* pcuFreqKernel,
    float3* pDetCenter,
    float3* pSrc,
    size_t nview,
    size_t nu,
    float du,
    float dx,
    float dz,
    float cutoffX,
    float cutoffZ,
    int filterType,
    hipStream_t& stream
)
{
    int filterLen = 2 * nu - 1;

    // fft plan
    hipfftHandle plan = 0;
    hipfftHandle planR2C = 0;
    hipfftComplex* pcuRamp = NULL;
    hipfftReal *pcuRealKernel = NULL;
    hipfftComplex* pRamp = NULL;
    hipfftComplex* pWindow = NULL;
    hipfftReal* pRealKernel = NULL;

    try
    {
        if (HIPFFT_SUCCESS != hipfftPlan1d(&plan, filterLen, HIPFFT_C2C, 1))
        {
            throw std::runtime_error("hipfftPlan1d failure");
        }
    
        if (HIPFFT_SUCCESS != hipfftSetStream(plan, stream))
        {
            throw std::runtime_error("cudaSetStream failure");
        }
    
        // RL kernel
        if (hipSuccess != hipMalloc(&pcuRamp, sizeof(hipfftComplex) * filterLen))
        {
            throw std::runtime_error("pcuRamp allocation error");
        }

        if (hipSuccess != hipMalloc(&pcuRealKernel, sizeof(hipfftReal) * filterLen))
        {
            throw std::runtime_error("pRealKernel allocation error");
        }

        pRealKernel = new hipfftReal [filterLen];
        pRamp = new hipfftComplex [filterLen];
        pWindow = new hipfftComplex [filterLen];

        // generate filter for each angle
        for (int iview = 0; iview < nview; iview++)
        {
            cout << iview << endl;

            // get the angle of the current projection
            // beta is the angle from x axis to the frequency plane
            float3 src = pSrc[iview];
            float3 det = pDetCenter[iview];
            float r = sqrtf((det.x - src.x) * (det.x - src.x) + (det.z - src.z) * (det.z - src.z));
            float sinDeg = (det.x - src.x) / r;
            float cosDeg = (src.z - det.z) / r;

            // a virtual detector at the frequency plane
            float vdu = du * cosDeg;

            // equispace
            for (int i = 0; i < filterLen; i++)
            {
                int k = i - (nu - 1);
                if (k == 0)
                {
                    pRamp[i].x = 1 / (4 * vdu * vdu);
                }
                else if (k % 2 != 0)
                {
                    pRamp[i].x = -1 / (PI * PI * k * k * vdu * vdu);
                }
                else
                {
                    pRamp[i].x = 0;
                }
                pRamp[i].y = 0;
            }

            hipMemcpyAsync(pcuRamp, pRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyHostToDevice, stream);
            hipStreamSynchronize(stream);
            hipfftExecC2C(plan, pcuRamp, pcuRamp, HIPFFT_FORWARD);
            hipMemcpyAsync(pRamp, pcuRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyDeviceToHost, stream);

            // weighting window in frequency domain
            switch(filterType)
            {
            case FILTER_HAMMING:
                // Hamming
                for (int i = 0; i < filterLen; i++)
                {
                    // fftshift
                    float wx = GetFrequency(i, filterLen, vdu, dx, cosDeg, cutoffX);
                    float wz = GetFrequency(i, filterLen, vdu, dz, sinDeg, cutoffZ);

                    pWindow[i].x = (0.54f + 0.46f * cosf(PI * wx / cutoffX)) * (0.54f + 0.46f * cosf(PI * wz / cutoffZ));
                    pWindow[i].y = 0;
                }
                break;
            case FILTER_HANN:
                for (int i = 0; i < filterLen; i++)
                {
                    float wx = GetFrequency(i, filterLen, vdu, dx, cosDeg, cutoffX);
                    float wz = GetFrequency(i, filterLen, vdu, dz, sinDeg, cutoffZ);

                    pWindow[i].x = (0.5f + 0.5f * cosf(PI * wx / cutoffX)) * (0.5f + 0.5f * cosf(PI * wz / cutoffZ));
                    pWindow[i].y = 0;
                }
                break;
            default:
                for (int i = 0; i < filterLen; i++)
                {
                    pWindow[i].x = 1;
                    pWindow[i].y = 0;
                }
            }

            // Apply window on the filter
            for (int i = 0; i < filterLen; i++)
            {
                float real = pRamp[i].x * pWindow[i].x - pRamp[i].y * pWindow[i].y;
                float imag = pRamp[i].x * pWindow[i].y + pRamp[i].y * pWindow[i].x;
                pRamp[i].x = real;
                pRamp[i].y = imag;
            }

            hipMemcpyAsync(pcuRamp, pRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyHostToDevice, stream);
            hipfftExecC2C(plan, pcuRamp, pcuRamp, HIPFFT_BACKWARD);
            hipMemcpyAsync(pRamp, pcuRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyDeviceToHost, stream);

            for (int i = 0; i < filterLen; i++)
            {
                pRealKernel[i] = pRamp[i].x / filterLen;
            }
            hipMemcpyAsync(pcuRealKernel, pRealKernel, sizeof(hipfftReal) * filterLen, hipMemcpyHostToDevice, stream);

            hipfftHandle planR2C;
            hipfftPlan1d(&planR2C, filterLen, HIPFFT_R2C, 1);
            hipfftSetStream(planR2C, stream);
            hipfftExecR2C(planR2C, pcuRealKernel, pcuFreqKernel + iview * nu);
        }
        

    }
    catch (exception &e)
    {
        if (pRamp != NULL) delete [] pRamp;
        if (pWindow != NULL) delete [] pWindow;
        if (pRealKernel != NULL) delete [] pRealKernel;
        if (pcuRamp != NULL) hipFree(pcuRamp);
        if (pcuRealKernel != NULL) hipFree(pcuRealKernel);
        if (plan != 0) hipfftDestroy(plan);
        if (planR2C != 0) hipfftDestroy(planR2C);

        ostringstream oss;
        oss << "GetRampTomo() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
        throw runtime_error(oss.str().c_str());
    }

    if (pRamp != NULL) delete [] pRamp;
    if (pWindow != NULL) delete [] pWindow;
    if (pRealKernel != NULL) delete [] pRealKernel;
    if (pcuRamp != NULL) hipFree(pcuRamp);
    if (pcuRealKernel != NULL) hipFree(pcuRealKernel);
    if (plan != 0) hipfftDestroy(plan);
    if (planR2C != 0) hipfftDestroy(planR2C);

}

void fbpTomo::Filter(float* pcuFPrj, const float* pcuPrj, const float* pcuDetCenter, const float* pcuSrc)
{
    // the filter is carried out for each different projection
    int filterLen = nu * 2 - 1;
    float* pcuPrjPad = NULL;
    hipfftComplex* pcuFreqPrj = NULL;
    hipfftComplex* pcuFilter = NULL;
    hipfftHandle plan = 0;
    hipfftHandle planInverse = 0;

    // cpu memory
    float3* pDetCenter = NULL;
    float3* pSrc = NULL;

    try
    {
        pDetCenter = new float3 [nview];
        pSrc = new float3 [nview];
        hipMemcpyAsync(pDetCenter, pcuDetCenter, sizeof(float3) * nview, hipMemcpyDeviceToHost, m_stream);
        hipMemcpyAsync(pSrc, pcuSrc, sizeof(float3) * nview, hipMemcpyDeviceToHost, m_stream);
        hipStreamSynchronize(m_stream);

        // projection
        if (hipSuccess != hipMalloc(&pcuPrjPad, sizeof(float) * filterLen * nv))
        {
            throw std::runtime_error("pcuPrjPad allocation failure");
        }

        // freq projection
        if (hipSuccess != hipMalloc(&pcuFreqPrj, sizeof(hipfftComplex) * nu * nv))
        {
            throw std::runtime_error("pcuFreqPrj allocation failure");
        }

        // filter
        if (hipSuccess != hipMalloc(&pcuFilter, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFilter allocation failure");
        }
        GetRampTomo(pcuFilter, pDetCenter, pSrc, nview, nu, du, dx, dz, cutoffX, cutoffZ, typeProjector, m_stream);

        // fft plan
        if (HIPFFT_SUCCESS != hipfftPlanMany(&plan, 1, &filterLen, NULL, 1, filterLen, NULL, 1, nu, HIPFFT_R2C, nv))
        {
            throw std::runtime_error("fft plan error");
        }
        hipfftSetStream(plan, m_stream);
        
        if (HIPFFT_SUCCESS != hipfftPlanMany(&planInverse, 1, &filterLen, NULL, 1, nu, NULL, 1, filterLen, HIPFFT_C2R, nv))
        {
            throw std::runtime_error("ifft plan error");
        }
        hipfftSetStream(plan, m_stream);

        // kernel threads and blocks
        dim3 threads(32, 16, 1);
        dim3 blocks(ceilf(nu / (float)threads.x), ceilf(nv / (float)threads.y), 1);

        for (int ib = 0; ib < nBatches; ib++)
        {
            for (int iview = 0; iview < nview; iview++)
            {
                // get the angle information
                float3 src = pSrc[iview];
                float3 det = pDetCenter[iview];
                float r = sqrtf((det.x - src.x) * (det.x - src.x) + (det.z - src.z) * (det.z - src.z));
                float cosDeg = (src.z - det.z) / r;
                // a virtual detector at the frequency plane
                float vdu = du * cosDeg;

                float scale = PI / nview * vdu / filterLen;

                // projection padding
                hipMemsetAsync(pcuPrjPad, 0, sizeof(float) * filterLen * nv, m_stream);
                hipMemcpy2DAsync(
                    pcuPrjPad,
                    filterLen * sizeof(float), 
                    pcuPrj + ib * nview * nu * nv + iview * nu * nv, nu * sizeof(float), 
                    nu * sizeof(float),
                    nv,
                    hipMemcpyDeviceToDevice,
                    m_stream
                );
                hipStreamSynchronize(m_stream);
                
                // filter
                hipfftExecR2C(plan, pcuPrjPad, pcuFreqPrj);
                FilterByFreqMultiply1D<<<blocks, threads, 0, m_stream>>>(pcuFreqPrj, pcuFreqPrj, pcuFilter, nu, nv);
                hipStreamSynchronize(m_stream);
                hipfftExecC2R(planInverse, pcuFreqPrj, pcuPrjPad);

                // post scaling
                Scale2D<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad + nu - 1, pcuPrjPad + nu - 1, scale, nu, nv, filterLen, filterLen
                );

                // get filtered projection
                hipMemcpy2DAsync(
                    pcuFPrj + ib * nview * nu * nv + iview * nu * nv,
                    nu * sizeof(float), 
                    pcuPrjPad + nu - 1,
                    filterLen * sizeof(float),
                    nu * sizeof(float),
                    nv,
                    hipMemcpyDeviceToDevice,
                    m_stream
                );
            }
        }
    }
    catch (exception &e)
    {
        if (plan != 0) hipfftDestroy(plan);
        if (planInverse != 0) hipfftDestroy(planInverse);
        if (pcuPrjPad != NULL) hipFree(pcuPrjPad);
        if (pcuFreqPrj != NULL) hipFree(pcuFreqPrj);
        if (pcuFilter != NULL) hipFree(pcuFilter);
        if (pDetCenter != NULL) delete [] pDetCenter;
        if (pSrc != NULL) delete [] pSrc;

        ostringstream oss;
        oss << "fbpTomo::Filter() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
        throw runtime_error(oss.str().c_str());
    }

    if (plan != 0) hipfftDestroy(plan);
    if (planInverse != 0) hipfftDestroy(planInverse);
    if (pcuPrjPad != NULL) hipFree(pcuPrjPad);
    if (pcuFreqPrj != NULL) hipFree(pcuFreqPrj);
    if (pcuFilter != NULL) hipFree(pcuFilter);
    if (pDetCenter != NULL) delete [] pDetCenter;
    if (pSrc != NULL) delete [] pSrc;

}

extern "C" int cupyFbpTomoFilter(
    float* fprj,
    const float* prj,
    const float* detCenter,
    const float* src,
    int nBatches,
    size_t nu,
    size_t nv,
    size_t nview,
    float du,
    float dx,
    float dz,
    int typeFilter = 0,
    float cutoffX = 1,
    float cutoffZ = 1
)
{
    try
	{
		fbpTomo projector;
		projector.Setup(
            nBatches, 512, 512, 512, dx, 1, dz, 0, 0, 0,
            nu, nv, nview, du, 1, 0, 0, 0, 0, typeFilter, cutoffX, cutoffZ
        );

		projector.Filter(fprj, prj, detCenter, src);

	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cupyFbpTomoFilter() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	return hipGetLastError();

}