#include "hip/hip_runtime.h"
/*
C-interface for distanceDriven3D.cu
*/

#include "distanceDriven.h"

#include <stdexcept>
#include <exception>
#include <sstream>
#include <iostream>
#include <fstream>

using namespace std;


void DistanceDrivenTomo::ProjectionTomo(const float* pcuImg, float* pcuPrj, const float* pcuDetCenter, const float* pcuSrc)
{
	// allocate buffers
	double* pcuAcc = NULL;
	int* pcuIviews = NULL;

	float3* pcuDetU = NULL;
	float3* pcuDetV = NULL;

	try
	{	
		if (typeProjector == 1)
		{
			if (hipSuccess != hipMalloc(&pcuAcc, sizeof(double) * (nx + 1) * (ny + 1) * nz))
			{
				throw runtime_error("pAcc allocation failed");
			}
		}

		if (hipSuccess != hipMalloc(&pcuDetU, sizeof(float3) * nview))
		{
			throw runtime_error("pDetU allocation failed");
		}

		if (hipSuccess != hipMalloc(&pcuDetV, sizeof(float3) * nview))
		{
			throw runtime_error("pDetV allocation failed");
		}

		if (hipSuccess != hipMalloc(&pcuIviews, sizeof(int) * nview))
		{
			throw runtime_error("cuIviews allocation failed");
		}

		// cuIviews should contain all angles
		int* iviews = new int [nview];
		for (int i = 0; i < nview; i++)
		{
			iviews[i] = i;
		}
		hipMemcpy(pcuIviews, iviews, sizeof(int) * nview, hipMemcpyHostToDevice);
		delete [] iviews;

		// pcuDetU should contain all (1,0,0)
		// pcuDetV should contain all (0,1,0)
		float3* pDetU = new float3 [nview];
		for (int i = 0; i < nview; i++)
		{
			pDetU[i] = make_float3(1, 0, 0);
		}
		hipMemcpy(pcuDetU, pDetU, sizeof(float3) * nview, hipMemcpyHostToDevice);
		delete [] pDetU;

		float3* pDetV = new float3 [nview];
		for (int i = 0; i < nview; i++)
		{
			pDetV[i] = make_float3(0, 1, 0);
		}
		hipMemcpy(pcuDetV, pDetV, sizeof(float3) * nview, hipMemcpyHostToDevice);
		delete [] pDetV;

		if (typeProjector == 1)
		{
			ProjectionTomoBranchless(
				pcuImg, pcuPrj, pcuDetCenter, pcuSrc, pcuAcc, pcuIviews, pcuDetU, pcuDetV
			);
		}
		else
		{
			ProjectionTomoBoxInt(
				pcuImg, pcuPrj, pcuDetCenter, pcuSrc, pcuIviews, pcuDetU, pcuDetV
			);
		}
	}
	catch (exception &e)
	{
		if (pcuAcc != NULL) hipFree(pcuAcc);
		if (pcuIviews != NULL) hipFree(pcuIviews);
		if (pcuDetU != NULL) hipFree(pcuDetU);
		if (pcuDetV != NULL) hipFree(pcuDetV);

		ostringstream oss;
		oss << "DistanceDrivenTomo::ProjectionTomo Error: " << e.what() 
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}

	if (pcuAcc != NULL) hipFree(pcuAcc);
	if (pcuIviews != NULL) hipFree(pcuIviews);
	if (pcuDetU != NULL) hipFree(pcuDetU);
	if (pcuDetV != NULL) hipFree(pcuDetV);

}

// no textures, use double-precision software interpolation
void DistanceDrivenTomo::BackprojectionTomo(float* pcuImg, const float* pcuPrj, const float* pcuDetCenter, const float* pcuSrc)
{
	// the backprojection is constrained to Cartesian coordinate for simplification, hence no detU / detV needed
	float* pcuWeightedPrjs = NULL;
	double* pcuAcc = NULL;
	int* pcuIviews = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuWeightedPrjs, sizeof(float) * nBatches * nu * nv * nview))
		{
			throw runtime_error("pWeightedPrjs allocation failed");
		}

		if (typeProjector == 1)
		{
			if (hipSuccess != hipMalloc(&pcuAcc, sizeof(double) * (nu + 1) * (nv + 1) * nview))
			{
				throw runtime_error("pAcc allocation failed");
			}
		}

		if (hipSuccess != hipMalloc(&pcuIviews, sizeof(int) * nview))
		{
			throw runtime_error("cuIviews allocation failed");
		}

		// cuIviews should contain all angles
		int* iviews = new int [nview];
		for (int i = 0; i < nview; i++)
		{
			iviews[i] = i;
		}
		hipMemcpy(pcuIviews, iviews, sizeof(int) * nview, hipMemcpyHostToDevice);
		delete [] iviews;

		if (typeProjector == 1)
		{
			BackprojectionTomoBranchless(
				pcuImg, pcuPrj, pcuDetCenter, pcuSrc, pcuWeightedPrjs, pcuAcc, pcuIviews
			);
		}
		else
		{
			BackprojectionTomoBoxInt(
				pcuImg, pcuPrj, pcuDetCenter, pcuSrc, pcuWeightedPrjs, pcuIviews
			);
		}

	}
	catch (exception &e)
	{
		if (pcuWeightedPrjs != NULL) hipFree(pcuWeightedPrjs);
		if (pcuAcc != NULL) hipFree(pcuAcc);
		if (pcuIviews != NULL) hipFree(pcuIviews);

		ostringstream oss;
		oss << "DistanceDrivenTomo::BackprojectionTomo Error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}

	if (pcuWeightedPrjs != NULL) hipFree(pcuWeightedPrjs);
	if (pcuAcc != NULL) hipFree(pcuAcc);
	if (pcuIviews != NULL) hipFree(pcuIviews);
}


// C interface
// typeProjector = 0: branchless mode, need double precision
// typeProjector = 1: box integral mode, single precision only
extern "C" int cDistanceDrivenTomoProjection(
	float* prj,
	const float* img,
	const float* detCenter,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v,
	int typeProjector
)
{
	float* pcuPrj = NULL;
	float* pcuImg = NULL;
	float* pcuDetCenter = NULL;
	float* pcuSrc = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nu * nv * nview * nBatches))
		{
			throw ("pcuPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nx * ny * nz * nBatches))
		{
			throw ("pcuImg allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetCenter, sizeof(float3) * nview))
		{
			throw ("pcuDetCenter allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuSrc, sizeof(float3) * nview))
		{
			throw ("pcuSrc allocation failed");
		}

		hipMemcpy(pcuImg, img, sizeof(float) * nx * ny * nz * nBatches, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetCenter, detCenter, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuSrc, src, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemset(pcuPrj, 0, sizeof(float) * nu * nview * nv * nBatches);

		DistanceDrivenTomo projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeProjector
		);

		projector.ProjectionTomo(pcuImg, pcuPrj, pcuDetCenter, pcuSrc);
		hipMemcpy(prj, pcuPrj, sizeof(float) * nu * nv * nview * nBatches, hipMemcpyDeviceToHost);
	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cDistanceDrivenTomoProjection() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	if (pcuPrj != NULL) hipFree(pcuPrj);
	if (pcuImg != NULL) hipFree(pcuImg);
	if (pcuDetCenter != NULL) hipFree(pcuDetCenter);
	if (pcuSrc != NULL) hipFree(pcuSrc);

	return hipGetLastError();

}

extern "C" int cupyDistanceDrivenTomoProjection(
	float* prj,
	const float* img,
	const float* detCenter,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v,
	int typeProjector
)
{
	try
	{
		DistanceDrivenTomo projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeProjector
		);

		projector.ProjectionTomo(img, prj, detCenter, src);
	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cupyDistanceDrivenTomoProjection() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	return hipGetLastError();
}


// C interface
extern "C" int cDistanceDrivenTomoBackprojection(
	float* img,
	const float* prj,
	const float* detCenter,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v,
	int typeProjector
)
{
	float* pcuPrj = NULL;
	float* pcuImg = NULL;
	float* pcuDetCenter = NULL;
	float* pcuSrc = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nu * nv * nview * nBatches))
		{
			throw ("pcuPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nx * ny * nz * nBatches))
		{
			throw ("pcuImg allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetCenter, sizeof(float3) * nview))
		{
			throw ("pcuDetCenter allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuSrc, sizeof(float3) * nview))
		{
			throw ("pcuSrc allocation failed");
		}

		hipMemset(pcuImg, 0, sizeof(float) * nx * ny * nz * nBatches);
		hipMemcpy(pcuDetCenter, detCenter, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuSrc, src, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuPrj, prj, sizeof(float) * nu * nv * nview * nBatches, hipMemcpyHostToDevice);

		DistanceDrivenTomo projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeProjector
		);

		projector.BackprojectionTomo(pcuImg, pcuPrj, pcuDetCenter, pcuSrc);
		hipMemcpy(img, pcuImg, sizeof(float) * nx * ny * nz * nBatches, hipMemcpyDeviceToHost);

	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cDistanceDrivenTomoProjection() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	if (pcuPrj != NULL) hipFree(pcuPrj);
	if (pcuImg != NULL) hipFree(pcuImg);
	if (pcuDetCenter != NULL) hipFree(pcuDetCenter);
	if (pcuSrc != NULL) hipFree(pcuSrc);

	return hipGetLastError();

}

extern "C" int cupyDistanceDrivenTomoBackprojection(
	float* img,
	const float* prj,
	const float* detCenter,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v,
	int typeProjector
)
{
	try
	{
		DistanceDrivenTomo projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeProjector
		);

		projector.BackprojectionTomo(img, prj, detCenter, src);

	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cDistanceDrivenTomoProjection() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	return hipGetLastError();

}
