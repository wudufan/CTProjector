#include "hip/hip_runtime.h"
#include "fbpMath.h"
#include "fbp.h"

#include <math.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <sstream>
#include <iostream>

using namespace std;

__global__ void ComplexMultiply2D(
    hipfftComplex* res, const hipfftComplex* op1, const hipfftComplex* op2, size_t nx, size_t ny
)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= nx || iy >= ny)
    {
        return;
    }

    size_t ind = iy * nx + ix;
    hipfftComplex val1 = op1[ind];
    hipfftComplex val2 = op2[ind];
    hipfftComplex val;

    val.x = val1.x * val2.x - val1.y * val2.y;
    val.y = val1.x * val2.y + val1.y * val2.x;

    res[ind] = val;
}

__global__ void FilterByFreqMultiply1D(
    hipfftComplex* res, const hipfftComplex* src, const hipfftComplex* filter, size_t nx, size_t ny
)
{
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= nx || iy >= ny)
    {
        return;
    }

    size_t ind = iy * nx + ix;
    hipfftComplex val1 = src[ind];
    hipfftComplex val2 = filter[ix];
    hipfftComplex val;

    val.x = val1.x * val2.x - val1.y * val2.y;
    val.y = val1.x * val2.y + val1.y * val2.x;

    res[ind] = val;
}

// The pcuPrjPad is of size [nview, nuPad]
__global__ void CopyPrjToPad(
    float* pcuPrjPad, const float* pcuPrj, int iv, size_t nu, size_t nuPad, size_t nv, size_t nview
)
{
    int iu = blockIdx.x * blockDim.x + threadIdx.x;
    int iview = blockIdx.y * blockDim.y + threadIdx.y;

    if (iu >= nu || iview >= nview)
    {
        return;
    }

    pcuPrjPad[iview * nuPad + iu] = pcuPrj[iview * nu * nv + iv * nu + iu];
}

__global__ void CopyPadToPrj(
    const float* pcuPrjPad, float* pcuPrj, int iv, size_t nu, size_t nuPad, size_t nv, size_t nview
)
{
    int iu = blockIdx.x * blockDim.x + threadIdx.x;
    int iview = blockIdx.y * blockDim.y + threadIdx.y;

    if (iu >= nu || iview >= nview)
    {
        return;
    }

    pcuPrj[iview * nu * nv + iv * nu + iu] = pcuPrjPad[iview * nuPad + iu];
}


void GetRamp(
    hipfftComplex* pcuFreqKernel,
    size_t nu,
    size_t nview,
    float da,
    int filterType,
    hipStream_t& stream,
    bool isEqualSpace
)
{
    int filterLen = 2 * nu - 1;

    // fft plan
    hipfftHandle plan = 0;
    hipfftHandle planR2C = 0;
    hipfftComplex* pcuRamp = NULL;
    hipfftReal *pcuRealKernel = NULL;
    hipfftComplex* pRamp = NULL;
    hipfftComplex* pWindow = NULL;
    hipfftReal* pRealKernel = NULL;

    try
    {
        if (HIPFFT_SUCCESS != hipfftPlan1d(&plan, filterLen, HIPFFT_C2C, 1))
        {
            throw std::runtime_error("hipfftPlan1d failure in GetRamp()");
        }
    
        if (HIPFFT_SUCCESS != hipfftSetStream(plan, stream))
        {
            throw std::runtime_error("cudaSetStream failure in GetRamp()");
        }
    
        // RL kernel
        if (hipSuccess != hipMalloc(&pcuRamp, sizeof(hipfftComplex) * filterLen))
        {
            throw std::runtime_error("pcuRamp allocation error in GetRamp()");
        }
        pRamp = new hipfftComplex [filterLen];
        if (isEqualSpace)
        {
            // equispace
            for (int i = 0; i < filterLen; i++)
            {
                int k = i - (nu - 1);
                if (k == 0)
                {
                    pRamp[i].x = 1 / (4 * da * da);
                }
                else if (k % 2 != 0)
                {
                    pRamp[i].x = -1 / (PI * PI * k * k * da * da);
                }
                else
                {
                    pRamp[i].x = 0;
                }
                pRamp[i].y = 0;
            }
        }
        else
        {
            // equiangular
            for (int i = 0; i < filterLen; i++)
            {
                int k = i - (nu - 1);
                if (k == 0)
                {
                    pRamp[i].x = 1 / (4 * da * da);
                }
                else if (k % 2 != 0)
                {
                    pRamp[i].x = -1 / (PI * PI * sinf(k*da) * sinf(k*da));
                }
                else
                {
                    pRamp[i].x = 0;
                }
                pRamp[i].y = 0;
            }
        }
    
        hipMemcpyAsync(pcuRamp, pRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyHostToDevice, stream);
        hipfftExecC2C(plan, pcuRamp, pcuRamp, HIPFFT_FORWARD);
        hipMemcpyAsync(pRamp, pcuRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyDeviceToHost, stream);
    
        // weighting window in frequency domain
        pWindow = new hipfftComplex [filterLen];
        switch(filterType)
        {
        case FILTER_HAMMING:
            // Hamming
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = 0.54f + 0.46f * cosf(2 * PI * i / (float)filterLen);
                pWindow[i].y = 0;
            }
            break;
        case FILTER_HANN:
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = 0.5f + 0.5f * cosf(2 * PI * i / (float)filterLen);
                pWindow[i].y = 0;
            }
            break;
        case FILTER_COSINE:
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = abs(cosf(PI * i / (float)filterLen));
                pWindow[i].y = 0;
            }
            break;
        default:
            for (int i = 0; i < filterLen; i++)
            {
                pWindow[i].x = 1;
                pWindow[i].y = 0;
            }
        }
    
        // Apply window on the filter
        for (int i = 0; i < filterLen; i++)
        {
            float real = pRamp[i].x * pWindow[i].x - pRamp[i].y * pWindow[i].y;
            float imag = pRamp[i].x * pWindow[i].y + pRamp[i].y * pWindow[i].x;
            pRamp[i].x = real;
            pRamp[i].y = imag;
        }
    
        hipMemcpyAsync(pcuRamp, pRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyHostToDevice, stream);
        hipfftExecC2C(plan, pcuRamp, pcuRamp, HIPFFT_BACKWARD);
        hipMemcpyAsync(pRamp, pcuRamp, sizeof(hipfftComplex) * filterLen, hipMemcpyDeviceToHost, stream);

        if (hipSuccess != hipMalloc(&pcuRealKernel, sizeof(hipfftReal) * filterLen))
        {
            throw std::runtime_error("pRealKernel allocation error in getRampEA");
        }
        hipfftReal* pRealKernel = new hipfftReal [filterLen];
        for (int i = 0; i < filterLen; i++)
        {
            pRealKernel[i] = pRamp[i].x / filterLen;
        }
        hipMemcpyAsync(pcuRealKernel, pRealKernel, sizeof(hipfftReal) * filterLen, hipMemcpyHostToDevice, stream);
    
        hipfftHandle planR2C;
        hipfftPlan1d(&planR2C, filterLen, HIPFFT_R2C, 1);
        hipfftSetStream(planR2C, stream);
        hipfftExecR2C(planR2C, pcuRealKernel, pcuFreqKernel);
        for (int i = 1; i < nview; i++)
        {
            hipMemcpyAsync(pcuFreqKernel + i * nu, pcuFreqKernel, sizeof(hipfftComplex) * nu, hipMemcpyDeviceToDevice, stream);
        }

    }
    catch (exception &e)
    {
        if (pRamp != NULL) delete [] pRamp;
        if (pWindow != NULL) delete [] pWindow;
        if (pRealKernel != NULL) delete [] pRealKernel;
        if (pcuRamp != NULL) hipFree(pcuRamp);
        if (pcuRealKernel != NULL) hipFree(pcuRealKernel);
        if (plan != 0) hipfftDestroy(plan);
        if (planR2C != 0) hipfftDestroy(planR2C);

        ostringstream oss;
        oss << "GetRamp() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
        throw runtime_error(oss.str().c_str());
    }

    if (pRamp != NULL) delete [] pRamp;
    if (pWindow != NULL) delete [] pWindow;
    if (pRealKernel != NULL) delete [] pRealKernel;
    if (pcuRamp != NULL) hipFree(pcuRamp);
    if (pcuRealKernel != NULL) hipFree(pcuRealKernel);
    if (plan != 0) hipfftDestroy(plan);
    if (planR2C != 0) hipfftDestroy(planR2C);

}