#include "hip/hip_runtime.h"
#include "siddon.h"
#include "projector.h"
#include "cudaMath.h"
#include "siddonCone.h"

#include <stdexcept>
#include <iostream>
#include <sstream>

using namespace std;

__global__ void SiddonConeProjectionArbitraryKernel(
	float* pPrj,
	const float* pImg,
	const float3* pDetCenter,
	const float3* pDetU,
	const float3* pDetV,
	const float3* pSrc,
	int nview,
	const Detector det,
	const Grid grid
)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iv = blockDim.y * blockIdx.y + threadIdx.y;
	int iview = blockDim.z * blockIdx.z + threadIdx.z;

	if (iu >= det.nu || iv >= det.nv || iview >= nview)
	{
		return;
	}

	float u = (iu - det.off_u - (det.nu - 1) / 2.0f) * det.du;
	float v = (iv - det.off_v - (det.nv - 1) / 2.0f) * det.dv;

	float3 src = pSrc[iview];
	float3 dst = UVToCart(u, v, pDetCenter[iview], pDetU[iview], pDetV[iview]);
	MoveSourceDstNearGrid(src, dst, grid);

	SiddonRayTracing(pPrj + iview * det.nu * det.nv + iv * det.nu + iu, pImg, src, dst, grid);

}

__global__ void SiddonConeBackprojectionArbitraryKernel(
	float* pImg,
	const float* pPrj,
	const float3* pDetCenter,
	const float3* pDetU,
	const float3* pDetV,
	const float3* pSrc,
	int nview,
	const Detector det,
	const Grid grid
)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iv = blockDim.y * blockIdx.y + threadIdx.y;
	int iview = blockDim.z * blockIdx.z + threadIdx.z;

	if (iu >= det.nu || iv >= det.nv || iview >= nview)
	{
		return;
	}

	float u = (iu - det.off_u - (det.nu - 1) / 2.0f) * det.du;
	float v = (iv - det.off_v - (det.nv - 1) / 2.0f) * det.dv;

	float3 src = pSrc[iview];
	float3 dst = UVToCart(u, v, pDetCenter[iview], pDetU[iview], pDetV[iview]);
	MoveSourceDstNearGrid(src, dst, grid);

	SiddonRayTracingTransposeAtomicAdd(pImg, pPrj[iview * det.nu * det.nv + iv * det.nu + iu], src, dst, grid);

}

void SiddonCone::ProjectionArbitrary(
	const float* pcuImg,
	float* pcuPrj,
	const float3* pcuDetCenter,
	const float3* pcuDetU,
	const float3* pcuDetV,
	const float3* pcuSrc
)
{
	dim3 threads, blocks;
	GetThreadsForXY(threads, blocks, nu, nv, nview);

	for (int ib = 0; ib < nBatches; ib++)
	{
		SiddonConeProjectionArbitraryKernel<<<blocks, threads, 0, m_stream>>>(
			pcuPrj + ib * nu * nv * nview,
			pcuImg + ib * nx * ny * nz,
			pcuDetCenter, 
			pcuDetU,
			pcuDetV, 
			pcuSrc,
			nview,
			MakeDetector(nu, nv, du, dv, off_u, off_v),
			MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz)
		);

		hipStreamSynchronize(m_stream);
	}

}

void SiddonCone::BackprojectionArbitrary(
	float* pcuImg,
	const float* pcuPrj,
	const float3* pcuDetCenter,
	const float3* pcuDetU,
	const float3* pcuDetV,
	const float3* pcuSrc
)
{
	dim3 threads, blocks;
	GetThreadsForXY(threads, blocks, nu, nv, nview);

	for (int ib = 0; ib < nBatches; ib++)
	{
		SiddonConeBackprojectionArbitraryKernel<<<blocks, threads, 0, m_stream>>>(
			pcuImg + ib * nx * ny * nz,
			pcuPrj + ib * nu * nv * nview,
			pcuDetCenter, 
			pcuDetU,
			pcuDetV, 
			pcuSrc,
			nview, 
			MakeDetector(nu, nv, du, dv, off_u, off_v),
			MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz)
		);

		hipStreamSynchronize(m_stream);
	}
}

extern "C" int cSiddonConeProjectionArbitrary(
	float* prj,
	const float* img,
	const float* detCenter,
	const float* detU,
	const float* detV,
	const float* src,
	size_t nBatches, 
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v
)
{
	float* pcuPrj = NULL;
	float* pcuImg = NULL;
	float3* pcuDetCenter = NULL;
	float3* pcuDetU = NULL;
	float3* pcuDetV = NULL;
	float3* pcuSrc = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nu * nv * nview * nBatches))
		{
			throw ("pcuPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nx * ny * nz * nBatches))
		{
			throw ("pcuImg allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetCenter, sizeof(float3) * nview))
		{
			throw ("pcuDetCenter allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetU, sizeof(float3) * nview))
		{
			throw ("pcuDetU allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetV, sizeof(float3) * nview))
		{
			throw ("pcuDetV allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuSrc, sizeof(float3) * nview))
		{
			throw ("pcuSrc allocation failed");
		}

		hipMemcpy(pcuImg, img, sizeof(float) * nx * ny * nz * nBatches, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetCenter, detCenter, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetU, detU, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetV, detV, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuSrc, src, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemset(pcuPrj, 0, sizeof(float) * nu * nview * nv * nBatches);

		SiddonCone projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, 0
		);

		projector.ProjectionArbitrary(pcuImg, pcuPrj, pcuDetCenter, pcuDetU, pcuDetV, pcuSrc);
		hipMemcpy(prj, pcuPrj, sizeof(float) * nu * nv * nview * nBatches, hipMemcpyDeviceToHost);
	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cSiddonConeProjectionArbitrary() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;

	}

	if (pcuPrj != NULL) hipFree(pcuPrj);
	if (pcuImg != NULL) hipFree(pcuImg);
	if (pcuDetCenter != NULL) hipFree(pcuDetCenter);
	if (pcuDetU != NULL) hipFree(pcuDetU);
	if (pcuDetV != NULL) hipFree(pcuDetV);
	if (pcuSrc != NULL) hipFree(pcuSrc);

	return hipGetLastError();

}

extern "C" int cupySiddonConeProjectionArbitrary(
	float* prj,
	const float* img,
	const float* detCenter,
	const float* detU,
	const float* detV,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v
)
{
	try
	{
		SiddonCone projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, 0
		);

		projector.ProjectionArbitrary(img, prj, (const float3*)detCenter, (const float3*)detU, (const float3*)detV, (const float3*)src);
	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cSiddonConeProjectionArbitrary() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;

	}

	return hipGetLastError();

}

extern "C" int cSiddonConeBackprojectionArbitrary(
	float* img,
	const float* prj,
	const float* detCenter,
	const float* detU,
	const float* detV,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v
)
{
	float* pcuPrj = NULL;
	float* pcuImg = NULL;
	float3* pcuDetCenter = NULL;
	float3* pcuDetU = NULL;
	float3* pcuDetV = NULL;
	float3* pcuSrc = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nu * nv * nview * nBatches))
		{
			throw ("pcuPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nx * ny * nz * nBatches))
		{
			throw ("pcuImg allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetCenter, sizeof(float3) * nview))
		{
			throw ("pcuDetCenter allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetU, sizeof(float3) * nview))
		{
			throw ("pcuDetU allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetV, sizeof(float3) * nview))
		{
			throw ("pcuDetV allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuSrc, sizeof(float3) * nview))
		{
			throw ("pcuSrc allocation failed");
		}

		hipMemcpy(pcuPrj, prj, sizeof(float) * nu * nv * nview * nBatches, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetCenter, detCenter, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetU, detU, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuDetV, detV, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemcpy(pcuSrc, src, sizeof(float3) * nview, hipMemcpyHostToDevice);
		hipMemset(pcuImg, 0, sizeof(float) * nx * ny * nz * nBatches);

		SiddonCone projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, 0
		);

		projector.BackprojectionArbitrary(pcuImg, pcuPrj, pcuDetCenter, pcuDetU, pcuDetV, pcuSrc);

		hipMemcpy(img, pcuImg, sizeof(float) * nx * ny * nz * nBatches, hipMemcpyDeviceToHost);
	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cSiddonConeBackprojectionArbitrary() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	if (pcuPrj != NULL) hipFree(pcuPrj);
	if (pcuImg != NULL) hipFree(pcuImg);
	if (pcuDetCenter != NULL) hipFree(pcuDetCenter);
	if (pcuDetU != NULL) hipFree(pcuDetU);
	if (pcuDetV != NULL) hipFree(pcuDetV);
	if (pcuSrc != NULL) hipFree(pcuSrc);

	return hipGetLastError();

}

extern "C" int cupySiddonConeBackprojectionArbitrary(
	float* img,
	const float* prj,
	const float* detCenter,
	const float* detU,
	const float* detV,
	const float* src,
	size_t nBatches,
	size_t nx,
	size_t ny,
	size_t nz,
	float dx,
	float dy,
	float dz,
	float cx,
	float cy,
	float cz,
	size_t nu,
	size_t nv,
	size_t nview,
	float du,
	float dv,
	float off_u,
	float off_v
)
{
	try
	{
		SiddonCone projector;
		projector.Setup(
			nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nv, nview, du, dv, off_u, off_v, 0, 0, 0
		);

		projector.BackprojectionArbitrary(img, prj, (const float3*)detCenter, (const float3*)detU, (const float3*)detV, (const float3*)src);
	}
	catch (exception& e)
	{
		ostringstream oss;
		oss << "cSiddonConeBackprojectionArbitrary() failed: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
	}

	return hipGetLastError();

}
