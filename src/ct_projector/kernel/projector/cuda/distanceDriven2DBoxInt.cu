#include "hip/hip_runtime.h"
/*
Distance driven with box integral.
It should have better precision.
*/

#include "distanceDriven.h"
#include "distanceDriven2DUtils.h"
#include "cudaMath.h"
#include "projector.h"

#include <stdexcept>
#include <exception>
#include <sstream>
#include <iostream>

using namespace std;

// Trace the distance driven forward projection
__device__ float DDFPTracing(
	const float* pImg,
	float2 src1,
	float2 src2,
	float2 dst1,
	float2 dst2,
	float z,
	float deg,
	float a,
	float cosDeg,
	float sinDeg,
	const Grid& grid
)
{
    // interpolation along z
    z = ClampFloat(z, 0, grid.nz);
    int iz = int(z);
    int iz1 = iz + 1;
    iz = Clamp(iz, 0, grid.nz);
    iz1 = Clamp(iz1, 0, grid.nz);

    float wz = z - iz;

	float val = 0;
	// calculate the intersection with at each y
	if (fabsf(cosDeg) > fabsf(sinDeg))
	{
		// calculate the intersection with at each y
		float r1 = (dst1.x - src1.x) / (dst1.y - src1.y);
		float r2 = (dst2.x - src2.x) / (dst2.y - src2.y);
		for (int iy = 0; iy < grid.ny; iy++)
		{
			float x1 = src1.x + r1 * (iy - src1.y);
			float x2 = src2.x + r2 * (iy - src2.y);

			if (x1 > x2) {
				float t = x1;
				x1 = x2;
				x2 = t;
			}

			val += (
                IntegralBoxX(pImg, x1, x2, iy, iz, grid.nx, grid.ny, grid.nz) * (1 - wz)
                + IntegralBoxX(pImg, x1, x2, iy, iz1, grid.nx, grid.ny, grid.nz) * wz
            ) / (x2 - x1);
		}

		// normalize by length
		val *= grid.dy / fabsf(__cosf(deg - a));
	}
	else
	{
		// calculate the intersection with at each x
		float r1 = (dst1.y - src1.y) / (dst1.x - src1.x);
		float r2 = (dst2.y - src2.y) / (dst2.x - src2.x);
		for (int ix = 0; ix < grid.nx; ix++)
		{
			float y1 = src1.y + r1 * (ix - src1.x);
			float y2 = src2.y + r2 * (ix - src2.x);

			if (y1 > y2) {
				float t = y1;
				y1 = y2;
				y2 = t;
			}

			val += (
				IntegralBoxY(pImg, y1, y2, ix, iz, grid.nx, grid.ny, grid.nz) * (1 - wz)
                + IntegralBoxY(pImg, y1, y2, ix, iz1, grid.nx, grid.ny, grid.nz) * wz
			) / (y2 - y1);
		}

		// normalize by length
		val *= grid.dx / fabsf(__sinf(deg - a));
	}

	return val;
}


/*
Distance driven parallel projection

pPrjs - projection of size [nview, nv, nu]
pAccX - accumulation of images along x, size [nz, ny, nx+1]
pAccY - accumulation of images along y, size [nz, ny+1, nx]
pDeg - projection angles, size [nview]
nviews - total view number
grid - image grid
det - detector information

*/
__global__ void DDFPParallelKernel(
	float* pPrjs,
	const float* pImg,
	const float* pDeg,
	size_t nview,
	const Grid grid,
	const Detector det
)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iv = blockDim.y * blockIdx.y + threadIdx.y;
	int iview = blockIdx.z * blockDim.z + threadIdx.z;

	if (iu >= det.nu || iv >= det.nv || iview >= nview)
	{
		return;
	}

	float deg = pDeg[iview];

	float cosDeg = __cosf(deg);
	float sinDeg = __sinf(deg);
	float u = (-(iu - (det.nu-1) / 2.0f) - det.off_u) * det.du;
	float z = (iv - (det.nv-1) / 2.0f + det.off_v) * det.dv;

	// a virtual dso to put the src and dst
	float dso = grid.nx * grid.dx + grid.ny * grid.dy;

	// calculate the coordinates of the detector cell's edges
	float2 src1 = make_float2(u - det.du / 2, -dso);
	float2 src2 = make_float2(u + det.du / 2, -dso);
	float2 dst1 = make_float2(u - det.du / 2, dso);
	float2 dst2 = make_float2(u + det.du / 2, dso);
	src1 = make_float2(src1.x * cosDeg - src1.y * sinDeg, src1.x * sinDeg + src1.y * cosDeg);
	src2 = make_float2(src2.x * cosDeg - src2.y * sinDeg, src2.x * sinDeg + src2.y * cosDeg);
	dst1 = make_float2(dst1.x * cosDeg - dst1.y * sinDeg, dst1.x * sinDeg + dst1.y * cosDeg);
	dst2 = make_float2(dst2.x * cosDeg - dst2.y * sinDeg, dst2.x * sinDeg + dst2.y * cosDeg);

	// convert to image coordinate
	src1 = PhysicsToImg(src1, grid);
	src2 = PhysicsToImg(src2, grid);
	dst1 = PhysicsToImg(dst1, grid);
	dst2 = PhysicsToImg(dst2, grid);
	z = (z - grid.cz) / grid.dz + grid.nz / 2.f - 0.5f;
	
	float val = DDFPTracing(
		pImg, src1, src2, dst1, dst2, z, deg, 0, cosDeg, sinDeg, grid
	);

	pPrjs[iview * det.nu * det.nv + iv * det.nu + iu] = val;

}


/*
Distance driven fanbeam projection

pPrjs - projection of size [nview, nv, nu]
pAccX - accumulation of images along x, size [nz, ny, nx+1]
pAccY - accumulation of images along y, size [nz, ny+1, nx]
pDeg - projection angles, size [nview]
nviews - total view number
grid - image grid
det - detector information
dsd - distance between source and detector
dso - distance between source and iso-center

*/
__global__ void DDFPFanKernel(
	float* pPrjs,
	const float* pImg,
	const float* pDeg,
	size_t nview,
	const Grid grid,
	const Detector det,
	float dsd,
	float dso
)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iv = blockDim.y * blockIdx.y + threadIdx.y;
	int iview = blockIdx.z * blockDim.z + threadIdx.z;

	if (iu >= det.nu || iv >= det.nv || iview >= nview)
	{
		return;
	}

	float deg = pDeg[iview];

	float cosDeg = __cosf(deg);
	float sinDeg = __sinf(deg);
	float a = (-(iu - (det.nu-1) / 2.0f) - det.off_u) * det.du;
	float z = (iv - (det.nv-1) / 2.0f + det.off_v) * det.dv;

	// calculate the coordinates of the detector cell's edges
	float2 src = make_float2(dso * sinDeg, -dso * cosDeg);
	float2 dst1 = make_float2(dsd * __sinf(a - det.du / 2), -dso + dsd * __cosf(a - det.du / 2));
	float2 dst2 = make_float2(dsd * __sinf(a + det.du / 2), -dso + dsd * __cosf(a + det.du / 2));
	dst1 = make_float2(dst1.x * cosDeg - dst1.y * sinDeg, dst1.x * sinDeg + dst1.y * cosDeg);
	dst2 = make_float2(dst2.x * cosDeg - dst2.y * sinDeg, dst2.x * sinDeg + dst2.y * cosDeg);

	// convert to image coordinate
	src = PhysicsToImg(src, grid);
	dst1 = PhysicsToImg(dst1, grid);
	dst2 = PhysicsToImg(dst2, grid);
	z = (z - grid.cz) / grid.dz + grid.nz / 2.f - 0.5f;
	
	float val = DDFPTracing(
		pImg, src, src, dst1, dst2, z, deg, a, cosDeg, sinDeg, grid
	);

	pPrjs[iview * det.nu * det.nv + iv * det.nu + iu] = val;

}


void DistanceDrivenFan::ProjectionBoxInt(const float* pcuImg, float* pcuPrj, const float* pcuDeg)
{
	try
	{
		Grid grid = MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz);
		Detector det = MakeDetector(nu, nv, du, dv, off_u, off_v);

		for (int ib = 0; ib < nBatches; ib++)
		{
			// step 2: interpolation
			dim3 threadDet, blockDet;
			GetThreadsForXZ(threadDet, blockDet, nu, nv, nview);
			DDFPFanKernel<<<blockDet, threadDet, 0, m_stream>>>(
				pcuPrj + ib * nu * nv * nview, pcuImg + ib * nx * ny * nz, pcuDeg, nview, grid, det, dsd, dso
			);
			hipStreamSynchronize(m_stream);
		}
	}
	catch (exception &e)
	{
		ostringstream oss;
		oss << "DistanceDrivenFan::Projection Error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}
}


void DistanceDrivenParallel::ProjectionBoxInt(const float* pcuImg, float* pcuPrj, const float* pcuDeg)
{
	try
	{
		Grid grid = MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz);
		Detector det = MakeDetector(nu, nv, du, dv, off_u, off_v);

		for (int ib = 0; ib < nBatches; ib++)
		{		
			dim3 threadDet, blockDet;
			GetThreadsForXZ(threadDet, blockDet, nu, nv, nview);
			DDFPParallelKernel<<<blockDet, threadDet, 0, m_stream>>>(
				pcuPrj + ib * nu * nv * nview, pcuImg + ib * nx * ny * nz, pcuDeg, nview, grid, det
			);
			hipStreamSynchronize(m_stream);
		}
	}
	catch (exception &e)
	{
		ostringstream oss;
		oss << "DistanceDrivenParallel::ProjectionBoxInt Error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}
}

///////////////////////////////////////////
// Backprojection
///////////////////////////////////////////

/*
Distance driven fanbeam backprojection

pImg - image of size (nz, ny, nx)
pPrj - projection of size (nview, nv, nu)
pDeg - projection angles, size [nview]
nviews - total view number
grid - image grid
det - detector information
dsd - distance between source and detector
dso - distance between source and iso-center
isFBP - if use FBP weighting for backprojection

*/
__global__ void DDBPFanKernel(
	float* pImg,
	const float* pPrj,
	const float* pDeg,
	size_t nview,
	const Grid grid,
	const Detector det,
	float dsd,
	float dso,
	bool isFBP
)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= grid.nx || iy >= grid.ny || iz >= grid.nz)
	{
		return;
	}

	float x = (ix - (grid.nx - 1) / 2.0f) * grid.dx;
	float x1 = x - grid.dx / 2.0f;
	float x2 = x + grid.dx / 2.0f;
	float y = (iy - (grid.ny - 1) / 2.0f) * grid.dy;
	float y1 = y - grid.dy / 2.0f;
	float y2 = y + grid.dy / 2.0f;
	float iv = (iz - (grid.nz - 1) / 2.0f) * grid.dz / det.dv + det.off_v + (det.nv - 1.0f) / 2.f;

	for (int iview = 0; iview < nview; iview++)
	{
		float deg = pDeg[iview];

		float cosDeg = __cosf(deg);
		float sinDeg = __sinf(deg);

		// For the u direction, the origin is the border of pixel 0
		// a1 and a2 are directly corresponding to the pixel-center coordinates of the accumulated pAcc
		float a1, a2;
		if (fabsf(cosDeg) > fabsf(sinDeg))
		{
			a1 = GetProjectionOnDetector(x1, y, dsd, dso, cosDeg, sinDeg);
			a2 = GetProjectionOnDetector(x2, y, dsd, dso, cosDeg, sinDeg);
		}
		else
		{
			a1 = GetProjectionOnDetector(x, y1, dsd, dso, cosDeg, sinDeg);
			a2 = GetProjectionOnDetector(x, y2, dsd, dso, cosDeg, sinDeg);
		}
		a1 = -(a1 / det.du + det.off_u) + det.nu / 2.0f;
		a2 = -(a2 / det.du + det.off_u) + det.nu / 2.0f;

		// make sure a1 < a2
		if (a1 > a2)
		{
			float t = a1;
			a1 = a2;
			a2 = t;
		}

		float val = IntegralBoxX(pPrj, a1, a2, iv, iview, det.nu, det.nv, nview) / (a2 - a1);

		if (isFBP)
		{
			pImg[iz * grid.nx * grid.ny + iy * grid.nx + ix] += val * GetFBPWeight(x, y, dsd, dso, cosDeg, sinDeg);
		}
		else
		{
			pImg[iz * grid.nx * grid.ny + iy * grid.nx + ix] += val;
		}

	}

}

/*
Distance driven parallel backprojection

pImg - image of size (nz, ny, nx)
pAcc - accumulation of pojection along u, of size (nview, nv, nu+1)
pDeg - projection angles, size [nview]
nviews - total view number
grid - image grid
det - detector information

*/
__global__ void DDBPParallelKernel(
	float* pImg,
	const float* pPrj,
	const float* pDeg,
	size_t nview,
	const Grid grid,
	const Detector det
)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= grid.nx || iy >= grid.ny || iz >= grid.nz)
	{
		return;
	}

	float x = (ix - (grid.nx - 1) / 2.0f) * grid.dx;
	float x1 = x - grid.dx / 2.0f;
	float x2 = x + grid.dx / 2.0f;
	float y = (iy - (grid.ny - 1) / 2.0f) * grid.dy;
	float y1 = y - grid.dy / 2.0f;
	float y2 = y + grid.dy / 2.0f;
	float iv = (iz - (grid.nz - 1) / 2.0f) * grid.dz / det.dv + det.off_v + (det.nv - 1.0f) / 2.f;

	for (int iview = 0; iview < nview; iview++)
	{
		float deg = pDeg[iview];

		float cosDeg = __cosf(deg);
		float sinDeg = __sinf(deg);

		// For the u direction, the origin is the border of pixel 0
		// u1 and u2 are directly corresponding to the pixel-center coordinates of the accumulated pAcc
		float u1, u2;
		if (fabsf(cosDeg) > fabsf(sinDeg))
		{
			u1 = GetProjectionOnDetectorParallel(x1, y, cosDeg, sinDeg);
			u2 = GetProjectionOnDetectorParallel(x2, y, cosDeg, sinDeg);
		}
		else
		{
			u1 = GetProjectionOnDetectorParallel(x, y1, cosDeg, sinDeg);
			u2 = GetProjectionOnDetectorParallel(x, y2, cosDeg, sinDeg);
		}
		u1 = -(u1 / det.du + det.off_u) + det.nu / 2.0f;
		u2 = -(u2 / det.du + det.off_u) + det.nu / 2.0f;

		// make sure a1 < a2
		if (u1 > u2)
		{
			float t = u1;
			u1 = u2;
			u2 = t;
		}

		float val = IntegralBoxX(pPrj, u1, u2, iv, iview, det.nu, det.nv, nview) / (u2 - u1);

		pImg[iz * grid.nx * grid.ny + iy * grid.nx + ix] += val;

	}

}

void DistanceDrivenFan::BackprojectionBoxInt(float* pcuImg, const float* pcuPrj, const float* pcuDeg)
{
	bool fbp = isFBP();

	try
	{
		Grid grid = MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz);
		Detector det = MakeDetector(nu, nv, du, dv, off_u, off_v);

		for (int ib = 0; ib < nBatches; ib++)
		{
			hipMemcpyAsync(pWeightedPrjs, pcuPrj + ib * nu * nv * nview, sizeof(float) * nu * nv * nview, hipMemcpyDeviceToDevice, m_stream);
			hipStreamSynchronize(m_stream);

			// pre-weight for iterative BP to make it conjugate to FP
			dim3 threadsDet, blocksDet;
			if (!fbp) // not FBP
			{
				GetThreadsForXZ(threadsDet, blocksDet, nu, nv, nview);
				PreweightBPKernel<<<blocksDet, threadsDet, 0, m_stream>>>(pWeightedPrjs, pcuDeg, nview, det, grid.dx, grid.dy);
				hipStreamSynchronize(m_stream);
			}

			// step 2: backprojection
			dim3 threads, blocks;
			GetThreadsForXY(threads, blocks, nx, ny, nz);
			DDBPFanKernel<<<blocks, threads, 0, m_stream>>>(
				pcuImg + ib * nx * ny * nz, pcuPrj + ib * nu * nv * nview, pcuDeg, nview, grid, det, dsd, dso, fbp
			);
			hipStreamSynchronize(m_stream);
		}
	}
	catch (exception &e)
	{
		ostringstream oss;
		oss << "DistanceDrivenFan::BackprojectionBoxInt error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}

}

void DistanceDrivenParallel::BackprojectionBoxInt(float* pcuImg, const float* pcuPrj, const float* pcuDeg)
{
	bool fbp = isFBP();

	try
	{
		Grid grid = MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz);
		Detector det = MakeDetector(nu, nv, du, dv, off_u, off_v);

		for (int ib = 0; ib < nBatches; ib++)
		{
			hipMemcpyAsync(pWeightedPrjs, pcuPrj + ib * nu * nv * nview, sizeof(float) * nu * nv * nview, hipMemcpyDeviceToDevice, m_stream);
			hipStreamSynchronize(m_stream);

			// pre-weight for iterative BP to make it conjugate to FP
			dim3 threadsDet, blocksDet;
			if (!fbp) // not FBP
			{
				GetThreadsForXZ(threadsDet, blocksDet, nu, nv, nview);
				PreweightBPParallelKernel<<<blocksDet, threadsDet, 0, m_stream>>>(pWeightedPrjs, pcuDeg, nview, det, grid.dx, grid.dy);
				hipStreamSynchronize(m_stream);
			}

			// backprojection
			dim3 threads, blocks;
			GetThreadsForXY(threads, blocks, nx, ny, nz);
			DDBPParallelKernel<<<blocks, threads, 0, m_stream>>>(
				pcuImg + ib * nx * ny * nz, pcuPrj + ib * nu * nv * nview, pcuDeg, nview, grid, det
			);
			hipStreamSynchronize(m_stream);
		}
	}
	catch (exception &e)
	{
		ostringstream oss;
		oss << "DistanceDrivenParallel::BackprojectionBoxInt error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}
}
