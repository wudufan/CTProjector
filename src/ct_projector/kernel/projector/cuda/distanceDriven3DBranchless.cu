#include "hip/hip_runtime.h"
/*
Distance driven 3D, branchless version.
It requires double precision of the GPU.
*/

#include "distanceDriven.h"
#include "distanceDriven3DUtils.h"
#include "cudaMath.h"
#include "siddon.h"

#include <stdexcept>
#include <exception>
#include <sstream>
#include <iostream>
#include <fstream>

using namespace std;
/* 
accumulate the pixel value along x axis
use double precision for dst because potentially a lot of pixels will be added
dst[0,y,z] = 0, dst[1,y,z] = src[0,y,z], dst[2,y,z] = src[0,y,z]+src[1,y,z], ...

dst - the buffer to receive the accumulation, of size (nx+1, ny+1, nz)
src - the original image, of size (nx, ny, nz)
nx,ny,nz - the dimension of the src image
*/
__global__ void AccumulateXYAlongXKernel(double* dst, const float* src, size_t nx, size_t ny, size_t nz)
{
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	int iz = blockDim.z * blockIdx.z + threadIdx.z;

	if (iy >= ny || iz >= nz)
	{
		return;
	}

	dst += iz * (nx + 1) * (ny + 1) + (iy + 1) * (nx + 1);	// skip iy == 0, which should always be zero in dst
	src += iz * nx * ny + iy * nx;
	dst[0] = 0;
	for (int ix = 0; ix < nx; ix++)
	{
		dst[ix + 1] = dst[ix] + src[ix];
	}
}

/*
accumulate the pixel value along y axis
use double precision for dst because potentially a lot of pixels will be added
the function should be called after AccumulateXYAlongXKernel. it directly accumulate inplace within the buffer

acc - the buffer for the accumulation, of size (nx+1, ny+1, nz)
nx,ny,nz - the dimension of the src image
*/
__global__ void AccumulateXYAlongYKernel(double* acc, size_t nx, size_t ny, size_t nz)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iz = blockDim.z * blockIdx.z + threadIdx.z;

	if (ix >= nx || iz >= nz)
	{
		return;
	}

	acc += iz * (nx + 1) * (ny + 1) + ix + 1;	// skip ix == 0, which should always be zero in dst
	
	for (int iy = 0; iy < ny; iy++)
	{
		acc[(iy + 1) * (nx + 1)] = acc[(iy + 1) * (nx + 1)] + acc[iy * (nx + 1)];
	}
}

/*
Distance driven projection. The projection should always be performed with z axis as the main axis.

iviews - the list of iview where DDFP is performed on the XY plane
nValidViews - length of iviews
*/
__global__ void DDFPConeKernelXY(
	float* pPrjs,
	const double* acc,
	const int* iviews,
	int nValidViews,
	int nview,
	const float3* pDetCenter,
	const float3* pDetU,
	const float3* pDetV,
	const float3* pSrc,
	const Grid grid,
	const Detector det
)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iv = blockDim.y * blockIdx.y + threadIdx.y;
	int ind = blockIdx.z * blockDim.z + threadIdx.z;

	if (iu >= det.nu || ind >= nValidViews || iv >= det.nv)
	{
		return;
	}

	int iview = iviews[ind];

	// coordinates of the center point of each edge of the detector's unit
	// dstx1, dstx2 are the edges at +- u along x axis
	// dsty1, dsty2 are the edges at +- v along y axis
	float u = (iu - det.off_u - (det.nu - 1) / 2.0f) * det.du;
	float v = (iv - det.off_v - (det.nv - 1) / 2.0f) * det.dv;
	float3 dstx1 = UVToCart(u - det.du / 2, v, pDetCenter[iview], pDetU[iview], pDetV[iview]);
	float3 dstx2 = UVToCart(u + det.du / 2, v, pDetCenter[iview], pDetU[iview], pDetV[iview]);
	float3 dsty1 = UVToCart(u, v - det.dv / 2, pDetCenter[iview], pDetU[iview], pDetV[iview]);
	float3 dsty2 = UVToCart(u, v + det.dv / 2, pDetCenter[iview], pDetU[iview], pDetV[iview]);

	float3 src = pSrc[iview];

	// convert to image coordinate
	src = PhysicsToImg(src, grid);
	dstx1 = PhysicsToImg(dstx1, grid);
	dstx2 = PhysicsToImg(dstx2, grid);
	dsty1 = PhysicsToImg(dsty1, grid);
	dsty2 = PhysicsToImg(dsty2, grid);

	// make sure dstx1.x < dstx2.x
	if (dstx1.x > dstx2.x)
	{
		float3 tmp = dstx1;
		dstx1 = dstx2;
		dstx2 = tmp;
	}

	// make sure dsty1.y < dsty2.y
	if (dsty1.y > dsty2.y)
	{
		float3 tmp = dsty1;
		dsty1 = dsty2;
		dsty2 = tmp;
	}

	float val = 0;
	float rx1 = (dstx1.x - src.x) / (dstx1.z - src.z);
	float rx2 = (dstx2.x - src.x) / (dstx2.z - src.z);
	float ry1 = (dsty1.y - src.y) / (dsty1.z - src.z);
	float ry2 = (dsty2.y - src.y) / (dsty2.z - src.z);

	// calculate intersection with each xy plane at different z
	for (int iz = 0; iz < grid.nz; iz++)
	{
		float x1 = src.x + rx1 * (iz - src.z);
		float x2 = src.x + rx2 * (iz - src.z);
		float y1 = src.y + ry1 * (iz - src.z);
		float y2 = src.y + ry2 * (iz - src.z);

		val += (
			InterpolateXY(acc, x2, y2, iz, grid.nx + 1, grid.ny + 1, grid.nz)
			+ InterpolateXY(acc, x1, y1, iz, grid.nx + 1, grid.ny + 1, grid.nz)
			- InterpolateXY(acc, x2, y1, iz, grid.nx + 1, grid.ny + 1, grid.nz)
			- InterpolateXY(acc, x1, y2, iz, grid.nx + 1, grid.ny + 1, grid.nz)
		) / ((x2 - x1) * (y2 - y1));

	}

	// normalize by length
	// use physics coordinate
	float3 dst = UVToCart(u, v, pDetCenter[iview], pDetU[iview], pDetV[iview]);
	src = pSrc[iview];
	val *= grid.dz / fabsf((src.z - dst.z)) * sqrtf(
		(src.z - dst.z) * (src.z - dst.z) + (src.y - dst.y) * (src.y-dst.y) + (src.x - dst.x) * (src.x - dst.x)
	);

	pPrjs[iview * det.nu * det.nv + iv * det.nu + iu] = val;

}

// no textures, use double-precision software interpolation
void DistanceDrivenTomo::ProjectionTomoBranchless(
	const float* pcuImg,
	float* pcuPrj,
	const float* pcuDetCenter,
	const float* pcuSrc,
	double* pcuAcc,
	const int* pcuIviews,
	const float3* pcuDetU,
	const float3* pcuDetV
)
{
	try
	{
		Grid grid = MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz);
		Detector det = MakeDetector(nu, nv, du, dv, off_u, off_v);

		dim3 threadX(1,32,16);
		dim3 blockX(1, ceilf(ny / 32.f), ceilf(nz / 16.f));
		dim3 threadY(32,1,16);
		dim3 blockY(ceilf(nx / 32.f), 1, ceilf(nz / 16.f));
		dim3 threads(32, 16, 1);
		dim3 blocks(ceilf(nu / 32.f), ceilf(nv / 16.f), nview);
		for (int ib = 0; ib < nBatches; ib++)
		{
			// step 1: calculate accumulated images
			// pAcc has the dimension in order (z, y, x)
			hipMemset(pcuAcc, 0, sizeof(double) * (nx + 1) * (ny + 1) * nz);
			AccumulateXYAlongXKernel<<<blockX, threadX>>>(pcuAcc, pcuImg + ib * nx * ny * nz , nx, ny, nz);
			hipDeviceSynchronize();
			AccumulateXYAlongYKernel<<<blockY, threadY>>>(pcuAcc, nx, ny, nz);
			hipDeviceSynchronize();

			// step 2: interpolation
			DDFPConeKernelXY<<<blocks, threads>>>(
				pcuPrj + ib * nu * nv * nview,
				pcuAcc + ib * (nx + 1) * (ny + 1) * nz,
				pcuIviews,
				nview,
				nview,
				(const float3*)pcuDetCenter,
				pcuDetU,
				pcuDetV,
				(const float3*)pcuSrc,
				grid,
				det
			);

			hipDeviceSynchronize();
		}

	}
	catch (exception &e)
	{
		ostringstream oss;
		oss << "DistanceDrivenTomo::ProjectionTomoBranchless Error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}
}

// dst has the dimension (batch, nview, nv+1, nu+1)
// src has the dimension (batch, nview, nv, nu)
__global__ void AccumulateUVAlongUKernel(double* dst, const float* src, size_t nu, size_t nv, size_t nview)
{
	int iv = blockDim.y * blockIdx.y + threadIdx.y;
	int iview = blockDim.z * blockIdx.z + threadIdx.z;

	if (iview >= nview || iv >= nv)
	{
		return;
	}

	dst += iview * (nu + 1) * (nv + 1) + (iv + 1) * (nu + 1);	// skip iv == 0, which should always be zero in dst
	src += iview * nu * nv + iv * nu;
	dst[0] = 0;
	for (int iu = 0; iu < nu; iu++)
	{
		dst[iu + 1] = dst[iu] + src[iu];
	}
}

// this kernel should be called right after AccumulateUVAlongUKernel to integrate along y axis,
// acc has the dimension (batch, nview, nv+1, nu+1)
__global__ void AccumulateUVAlongVKernel(double* acc, size_t nu, size_t nv, size_t nview)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iview = blockDim.z * blockIdx.z + threadIdx.z;

	if (iu >= nu || iview >= nview)
	{
		return;
	}

	acc += iview * (nu + 1) * (nv + 1) + iu + 1;	// skip iu == 0, which should always be zero in dst
	for (int iv = 0; iv < nv; iv++)
	{
		acc[(iv + 1) * (nu + 1)] = acc[(iv + 1) * (nu + 1)] + acc[iv * (nu + 1)];
	}
}

// BP when detector aligns with the cartesian coordinate
__global__ void DDBPConeCartKernelXY(
	float* pImg,
	const double* acc,
	const int* iviews,
	size_t nValidViews,
	size_t nview,
	const float3* pDetCenter,
	const float3* pSrc,
	const Grid grid,
	const Detector det
)
{
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= grid.nx || iy >= grid.ny || iz >= grid.nz)
	{
		return;
	}

	float x = (ix - (grid.nx - 1) / 2.0f) * grid.dx + grid.cx;
	float y = (iy - (grid.ny - 1) / 2.0f) * grid.dy + grid.cy;
	float z = (iz - (grid.nz - 1) / 2.0f) * grid.dz + grid.cz;

	float val = 0;
	for (int ind = 0; ind < nValidViews; ind++)
	{
		int iview = iviews[ind];
		float3 src = pSrc[iview];
		float3 detCenter = pDetCenter[iview];

		float u1 = ProjectConeToDetCart(make_float3(x - grid.dx / 2, y, z), detCenter, src, det).x;
		float u2 = ProjectConeToDetCart(make_float3(x + grid.dx / 2, y, z), detCenter, src, det).x;
		float v1 = ProjectConeToDetCart(make_float3(x, y - grid.dy / 2, z), detCenter, src, det).y;
		float v2 = ProjectConeToDetCart(make_float3(x, y + grid.dy / 2, z), detCenter, src, det).y;


		val += (
			InterpolateXY(acc, u2, v2, iview, det.nu + 1, det.nv + 1, nview)
			- InterpolateXY(acc, u2, v1, iview, det.nu + 1, det.nv + 1, nview)
			+ InterpolateXY(acc, u1, v1, iview, det.nu + 1, det.nv + 1, nview)
			- InterpolateXY(acc, u1, v2, iview, det.nu + 1, det.nv + 1, nview)
		) / ((u2 - u1) * (v2 - v1));

		// pImg[iz * grid.nx * grid.ny + iy * grid.nx + ix] = z - src.z;
		
	}

	pImg[iz * grid.nx * grid.ny + iy * grid.nx + ix] = val;

}

// no textures, use double-precision software interpolation
void DistanceDrivenTomo::BackprojectionTomoBranchless(
	float* pcuImg,
	const float* pcuPrj,
	const float* pcuDetCenter,
	const float* pcuSrc,
	float* pcuWeightedPrjs,
	double* pcuAcc,
	const int* pcuIviews
)
{
	try
	{
		Grid grid = MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz);
		Detector det = MakeDetector(nu, nv, du, dv, off_u, off_v);

		// step 0: preweight the projections for ray intersection length
		dim3 threadUV(32, 16, 1);
		dim3 blockUV(ceilf(nu / 32.f), ceilf(nv / 16.f), 1);
		hipMemcpy(pcuWeightedPrjs, pcuPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyDeviceToDevice);
		for (int ib = 0; ib < nBatches; ib++)
		{
			PreweightBPCartKernelXY<<<blockUV, threadUV>>>(
				pcuWeightedPrjs + ib * nu * nv * nview,
				pcuIviews,
				nview,
				nview,
				(const float3*)pcuDetCenter,
				(const float3*)pcuSrc,
				grid.dz,
				det
			);
		}
		hipDeviceSynchronize();

		dim3 threadU(1, 32, 4);
		dim3 blockU(1, ceilf(nv / 32.f), ceilf(nview / 4.f));
		dim3 threadV(32, 1, 4);
		dim3 blockV(ceilf(nu / 32.f), 1, ceilf(nview / 4.f));
		dim3 threadImg(32, 16, 1);
		dim3 blockImg(ceilf(nx / 32.f), ceilf(ny / 16.f), nz);
		for (int ib = 0; ib < nBatches; ib++)
		{
			hipMemset(pcuAcc, 0, sizeof(double) * (nu + 1) * (nv + 1) * nview);

			// step 1: calculate accumulated projections
			AccumulateUVAlongUKernel<<<blockU, threadU>>>(pcuAcc, pcuWeightedPrjs + ib * nu * nv * nview, nu, nv, nview);
			hipDeviceSynchronize();
			AccumulateUVAlongVKernel<<<blockV, threadV>>>(pcuAcc, nu, nv, nview);
			hipDeviceSynchronize();

			// step 2: interpolation
			DDBPConeCartKernelXY<<<blockImg, threadImg>>>(
				pcuImg + ib * nx * ny * nz,
				pcuAcc,
				pcuIviews,
				nview,
				nview,
				(const float3*)pcuDetCenter,
				(const float3*)pcuSrc,
				grid,
				det
			);
			hipDeviceSynchronize();
		}

	}
	catch (exception &e)
	{
		ostringstream oss;
		oss << "DistanceDrivenTomo::BackprojectionTomoBranchless Error: " << e.what()
			<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw oss.str().c_str();
	}
}
