#include "hip/hip_runtime.h"
#include "fbp.h"
#include "cudaMath.h"
#include "fbpMath.h"

#include <math.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <sstream>
#include <iostream>

using namespace std;


void fbpParallel::Filter(float* pcuFPrj, const float* pcuPrj)
{
	// the filter is carried out for each different v
	int filterLen = nu * 2 - 1;

	float* pcuPrjPad = NULL;                // projection
	hipfftComplex* pcuFreqPrj = NULL;        // freq projection
	hipfftComplex* pcuFilter = NULL;         // filter
	hipfftHandle plan;                       // fft plan
    hipfftHandle planInverse;

    try
    {
        if (hipSuccess != hipMalloc(&pcuPrjPad, sizeof(float) * filterLen * nview))
        {
            throw std::runtime_error("pcuPrjPad allocation failure in fbpParallel::Filter");
        }

        if (hipSuccess != hipMalloc(&pcuFreqPrj, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFreqPrj allocation failure in fbpParallel::Filter");
        }

        if (hipSuccess != hipMalloc(&pcuFilter, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFilter allocation failure in fbpParallel::Filter");
        }
        GetRamp(pcuFilter, nu, nview, du, typeProjector, m_stream, true);

        // no weighting for parallel filtering
        if (HIPFFT_SUCCESS != hipfftPlanMany(&plan, 1, &filterLen, NULL, 1, filterLen, NULL, 1, nu, HIPFFT_R2C, nview))
        {
            throw std::runtime_error("fft plan error in fan3D::Filter");
        }
        hipfftSetStream(plan, m_stream);

        if (HIPFFT_SUCCESS != hipfftPlanMany(&planInverse, 1, &filterLen, NULL, 1, nu, NULL, 1, filterLen, HIPFFT_C2R, nview))
        {
            throw std::runtime_error("ifft plan error in fan3D::Filter");
        }
        hipfftSetStream(plan, m_stream);

        // kernel threads and blocks
        dim3 threads(32, 32, 1);
        dim3 blocks(ceilf(nu / (float)threads.x), ceilf(nview / (float)threads.y), 1);
        float scale = PI / nview * du / filterLen;

        for (int ib = 0; ib < nBatches; ib++)
        {
            for (int iv = 0; iv < nv; iv++)
            {
                hipMemsetAsync(pcuPrjPad, 0, sizeof(float) * filterLen * nview, m_stream);
                CopyPrjToPad<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad, pcuPrj + ib * nu * nv * nview, iv, nu, filterLen, nv, nview
                );

                // no pre weighting for parallel filtering
                hipDeviceSynchronize();

                hipfftExecR2C(plan, pcuPrjPad, pcuFreqPrj);
                ComplexMultiply2D<<<blocks, threads, 0, m_stream>>>(pcuFreqPrj, pcuFreqPrj, pcuFilter, nu, nview);
                hipDeviceSynchronize();
                hipfftExecC2R(planInverse, pcuFreqPrj, pcuPrjPad);

                // post scaling
                Scale2D<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad + nu - 1, pcuPrjPad + nu - 1, scale, nu, nview, filterLen, filterLen
                );

                CopyPadToPrj<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad + nu - 1, pcuFPrj + ib * nu * nv * nview, iv, nu, filterLen, nv, nview
                );
            }
        }
    }
    catch(const std::exception& e)
    {   
        if (plan != 0) hipfftDestroy(plan);
        if (planInverse != 0) hipfftDestroy(planInverse);
        if (pcuPrjPad != NULL) hipFree(pcuPrjPad);
        if (pcuFreqPrj != NULL) hipFree(pcuFreqPrj);
        if (pcuFilter != NULL) hipFree(pcuFilter);

        std::ostringstream oss;
        oss << "fbpParallel::Filter() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
        std::cerr << oss.str() << std::endl;
        throw std::runtime_error(oss.str().c_str());
    }

	hipfftDestroy(plan);
	hipfftDestroy(planInverse);
	hipFree(pcuPrjPad);
	hipFree(pcuFreqPrj);
	hipFree(pcuFilter);
}

extern "C" int cupyfbpParallelFilter(
    float* pFPrj,
    const float* pPrj,
    int nBatches, 
    size_t nu,
    size_t nv,
    size_t nview,
    float du,
    float dv,
    float off_u,
    float off_v,
	int typeFilter = 0
) {
    fbpParallel filter;
	filter.Setup(
        nBatches, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeFilter
    );

    try
	{
        hipMemset(pFPrj, 0, sizeof(float) * nBatches * nu * nv * nview);
		filter.Filter(pFPrj, pPrj);
	}
	catch (std::exception &e)
	{
		std::ostringstream oss;
		oss << "cFilterParallelFilter() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
		std::cerr << oss.str() << std::endl;
	}

    return hipGetLastError();
}

extern "C" int cfbpParallelFilter(
    float* pFPrj,
    const float* pPrj,
    int nBatches, 
    size_t nu,
    size_t nv,
    size_t nview,
    float du,
    float dv,
    float off_u,
    float off_v,
	int typeFilter = 0
)
{
	fbpParallel filter;
	filter.Setup(
        nBatches, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeFilter
    );
	float* pcuFPrj = NULL;
	float* pcuPrj = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuFPrj, sizeof(float) * nBatches * nu * nv * nview))
		{
			throw std::runtime_error("pcuFPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nBatches * nu * nv * nview))
		{
			throw std::runtime_error("pcuPrj allocation failed");
		}
		hipMemset(pcuFPrj, 0, sizeof(float) * nBatches * nu * nv * nview);
		hipMemcpy(pcuPrj, pPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyHostToDevice);

		filter.Filter(pcuFPrj, pcuPrj);

        hipMemcpy(pFPrj, pcuFPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyDeviceToHost);
	}
	catch (std::exception &e)
	{
		std::ostringstream oss;
		oss << "cFilterParallelFilter() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
		std::cerr << oss.str() << std::endl;
	}

	if (pcuFPrj != NULL) hipFree(pcuFPrj);
    if (pcuPrj != NULL) hipFree(pcuPrj);

    return hipGetLastError();

}

const static int nzBatch = 5;
__global__ void bpParallelKernel3D(
    float* pImg,
    const float* prj,
    const float* pDeg,
    size_t nview,
    const Grid grid,
    const Detector det
)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int izBatch = blockIdx.z * blockDim.z + threadIdx.z;

	if (ix >= grid.nx || iy >= grid.ny || izBatch * nzBatch >= grid.nz)
	{
		return;
	}

    // the image coordinates here has the lower left corner of the first pixel defined as (0,0,0)
    // because (ix,iy,iz) are based on the centers of the pixels, so an offset of 0.5 should be added. 
    register float3 pt = ImgToPhysics(make_float3(ix + 0.5f, iy + 0.5f, izBatch * nzBatch + 0.5f), grid);

	register float val[nzBatch] = {0};
	register float cosDeg, sinDeg, pu, pv;
	for (int iview = 0; iview < nview; iview++)
	{
		cosDeg = __cosf(pDeg[iview]);
		sinDeg = __sinf(pDeg[iview]);
		pu =  pt.x * cosDeg + pt.y * sinDeg;

		pu = -(pu / det.du + det.off_u) + (det.nu - 1.0f) / 2.0f;

#pragma unroll
		for (int iz = 0; iz < nzBatch; iz++)
		{
			pv = (pt.z + iz * grid.dz) / det.dv + det.off_v + (det.nv - 1.0f) / 2.f;

            val[iz] += InterpolateXY(prj, pu, pv, iview, det.nu, det.nv, nview, true);
		}

	}
#pragma unroll
	for (int iz = 0; iz < nzBatch; iz++)
	{
		if (iz + izBatch * nzBatch < grid.nz)
		{
			pImg[(iz + izBatch * nzBatch) * grid.nx * grid.ny + iy * grid.nx + ix] += val[iz];
		}
	}

}

void fbpParallel::Backprojection(float* pcuImg, const float* pcuPrj, const float* pcuDeg)
{
    dim3 threads(32, 16, 1);
	dim3 blocks(ceilf(nx / (float)threads.x), ceilf(ny / (float)threads.y), ceilf(nz / (float)nzBatch));

	for (int ib = 0; ib < nBatches; ib++)
	{
        bpParallelKernel3D<<<blocks, threads, 0, m_stream>>>(
            pcuImg + ib * nx * ny * nz,
            pcuPrj + ib * nu * nv * nview,
            pcuDeg, 
            nview,
            MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz),
            MakeDetector(nu, nv, du, dv, off_u, off_v)
        );
        hipDeviceSynchronize();
	}
}


extern "C" int cfbpParallelBackprojection(
    float* pImg,
    const float* pPrj,
    const float* pDeg,
    size_t nBatches, 
    size_t nx,
    size_t ny,
    size_t nz,
    float dx,
    float dy,
    float dz,
    float cx,
    float cy,
    float cz,
    size_t nu,
    size_t nv,
    size_t nview,
    float du,
    float dv,
    float off_u,
    float off_v,
    int typeProjector = 0
)
{
    fbpParallel projector;
    projector.Setup(
        nBatches, nx, ny, nz, dx, dy, dz, cx, cy, cz,
        nu, nv, nview, du, dv, off_u, off_v, 1000, 500, typeProjector
    );

    float* pcuImg = NULL;
    float* pcuPrj = NULL;
    float* pcuDeg = NULL;
    try
    {
        if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nBatches * nx * ny * nz))
        {
            throw runtime_error("pcuImg allocation failed");
        }

        if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nBatches * nu * nview * nv))
        {
            throw runtime_error("pcuPrj allocation failed");
        }

        if (hipSuccess != hipMalloc(&pcuDeg, sizeof(float) * nview))
        {
            throw runtime_error("pcuDeg allocation failed");
        }

        hipMemcpy(pcuPrj, pPrj, sizeof(float) * nBatches * nu * nview * nv, hipMemcpyHostToDevice);
        hipMemcpy(pcuDeg, pDeg, sizeof(float) * nview, hipMemcpyHostToDevice);
        hipMemset(pcuImg, 0, sizeof(float) * nBatches * nx * ny * nz);

        projector.Backprojection(pcuImg, pcuPrj, pcuDeg);
        hipMemcpy(pImg, pcuImg, sizeof(float) * nBatches * nx * ny * nz, hipMemcpyDeviceToHost);

    }
    catch (exception &e)
    {
        ostringstream oss;
        oss << "cfbpParallelBackprojection() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
        cerr << oss.str() << endl;
    }

    if (pcuImg != NULL) hipFree(pcuImg);
    if (pcuPrj != NULL) hipFree(pcuPrj);
    if (pcuDeg != NULL) hipFree(pcuDeg);

    return hipGetLastError();
}
