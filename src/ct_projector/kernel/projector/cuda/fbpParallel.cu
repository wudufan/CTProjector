#include "hip/hip_runtime.h"
#include "fbp.h"
#include "cudaMath.h"
#include "fbpMath.h"

#include <math.h>
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <sstream>
#include <iostream>


void fbpParallel::Filter(float* pcuFPrj, const float* pcuPrj)
{
	// the filter is carried out for each different v
	int filterLen = nu * 2 - 1;

	float* pcuPrjPad = NULL;                // projection
	hipfftComplex* pcuFreqPrj = NULL;        // freq projection
	hipfftComplex* pcuFilter = NULL;         // filter
	hipfftHandle plan;                       // fft plan
    hipfftHandle planInverse;

    try
    {
        if (hipSuccess != hipMalloc(&pcuPrjPad, sizeof(float) * filterLen * nview))
        {
            throw std::runtime_error("pcuPrjPad allocation failure in fbpParallel::Filter");
        }

        if (hipSuccess != hipMalloc(&pcuFreqPrj, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFreqPrj allocation failure in fbpParallel::Filter");
        }

        if (hipSuccess != hipMalloc(&pcuFilter, sizeof(hipfftComplex) * nu * nview))
        {
            throw std::runtime_error("pcuFilter allocation failure in fbpParallel::Filter");
        }
        GetRamp(pcuFilter, nu, nview, du, typeProjector, m_stream, true);

        // no weighting for parallel filtering
        if (HIPFFT_SUCCESS != hipfftPlanMany(&plan, 1, &filterLen, NULL, 1, filterLen, NULL, 1, nu, HIPFFT_R2C, nview))
        {
            throw std::runtime_error("fft plan error in fan3D::Filter");
        }
        hipfftSetStream(plan, m_stream);

        if (HIPFFT_SUCCESS != hipfftPlanMany(&planInverse, 1, &filterLen, NULL, 1, nu, NULL, 1, filterLen, HIPFFT_C2R, nview))
        {
            throw std::runtime_error("ifft plan error in fan3D::Filter");
        }
        hipfftSetStream(plan, m_stream);

        // kernel threads and blocks
        dim3 threads(32, 32, 1);
        dim3 blocks(ceilf(nu / (float)threads.x), ceilf(nview / (float)threads.y), 1);
        float scale = PI / nview * du / filterLen;

        for (int ib = 0; ib < nBatches; ib++)
        {
            for (int iv = 0; iv < nv; iv++)
            {
                hipMemsetAsync(pcuPrjPad, 0, sizeof(float) * filterLen * nview, m_stream);
                CopyPrjToPad<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad, pcuPrj + ib * nu * nv * nview, iv, nu, filterLen, nv, nview
                );

                // no pre weighting for parallel filtering
                hipDeviceSynchronize();

                hipfftExecR2C(plan, pcuPrjPad, pcuFreqPrj);
                ComplexMultiply2D<<<blocks, threads, 0, m_stream>>>(pcuFreqPrj, pcuFreqPrj, pcuFilter, nu, nview);
                hipDeviceSynchronize();
                hipfftExecC2R(planInverse, pcuFreqPrj, pcuPrjPad);

                // post scaling
                Scale2D<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad + nu - 1, pcuPrjPad + nu - 1, scale, nu, nview, filterLen, filterLen
                );

                CopyPadToPrj<<<blocks, threads, 0, m_stream>>>(
                    pcuPrjPad + nu - 1, pcuFPrj + ib * nu * nv * nview, iv, nu, filterLen, nv, nview
                );
            }
        }
    }
    catch(const std::exception& e)
    {   
        if (plan != 0) hipfftDestroy(plan);
        if (planInverse != 0) hipfftDestroy(planInverse);
        if (pcuPrjPad != NULL) hipFree(pcuPrjPad);
        if (pcuFreqPrj != NULL) hipFree(pcuFreqPrj);
        if (pcuFilter != NULL) hipFree(pcuFilter);

        std::ostringstream oss;
        oss << "fbpParallel::Filter() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
        std::cerr << oss.str() << std::endl;
        throw std::runtime_error(oss.str().c_str());
    }

	hipfftDestroy(plan);
	hipfftDestroy(planInverse);
	hipFree(pcuPrjPad);
	hipFree(pcuFreqPrj);
	hipFree(pcuFilter);
}

extern "C" int cfbpParallelFilter(
    float* pFPrj,
    const float* pPrj,
    int nBatches, 
    size_t nu,
    size_t nv,
    size_t nview,
    float du,
    float dv,
    float off_u,
    float off_v,
	int typeFilter = 0
)
{
	fbpParallel filter;
	filter.Setup(
        nBatches, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        nu, nv, nview, du, dv, off_u, off_v, 0, 0, typeFilter
    );
	float* pcuFPrj = NULL;
	float* pcuPrj = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuFPrj, sizeof(float) * nBatches * nu * nv * nview))
		{
			throw std::runtime_error("pcuFPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nBatches * nu * nv * nview))
		{
			throw std::runtime_error("pcuPrj allocation failed");
		}
		hipMemset(pcuFPrj, 0, sizeof(float) * nBatches * nu * nv * nview * nv);
		hipMemcpy(pcuPrj, pPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyHostToDevice);

		filter.Filter(pcuFPrj, pcuPrj);

        hipMemcpy(pFPrj, pcuFPrj, sizeof(float) * nBatches * nu * nv * nview, hipMemcpyDeviceToHost);
	}
	catch (std::exception &e)
	{
		std::ostringstream oss;
		oss << "cFilterParallelFilter() failed: " << e.what()
            << "(" << hipGetErrorString(hipGetLastError()) << ")";
		std::cerr << oss.str() << std::endl;
	}

	if (pcuFPrj != NULL) hipFree(pcuFPrj);
    if (pcuPrj != NULL) hipFree(pcuPrj);

    return hipGetLastError();

}